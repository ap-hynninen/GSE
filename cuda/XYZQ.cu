#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include "cuda_utils.h"
#include "gpu_utils.h"
#include "XYZQ.h"

//
// XYZQ class method definitions
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//
//

//
// Copies x, y, z coordinates into xyzq -array
//
__global__ void set_xyz_kernel(const int ncoord,
			       const double* __restrict__ x,
			       const double* __restrict__ y,
			       const double* __restrict__ z,
			       float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    xyzq[tid].x = x[tid];
    xyzq[tid].y = y[tid];
    xyzq[tid].z = z[tid];
  }
}

//
// Copies (x, y, z, q) into xyzq -array
//
__global__ void set_xyzq_kernel(const int ncoord,
				const double* __restrict__ x,
				const double* __restrict__ y,
				const double* __restrict__ z,
				const float* __restrict__ q,
				float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    xyzq_val.x = x[tid];
    xyzq_val.y = y[tid];
    xyzq_val.z = z[tid];
    xyzq_val.w = q[tid];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z, q) into xyzq -array and also shifts (x, y, z)
//
__global__ void set_xyzq_shift_kernel(const int ncoord,
				      const double* __restrict__ x,
				      const double* __restrict__ y,
				      const double* __restrict__ z,
				      const float* __restrict__ q,
				      const int* __restrict__ loc2glo,
				      const float3* __restrict__ xyz_shift,
				      const double boxx, const double boxy, const double boxz,
				      float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(x[tid] + ((double)shift.x)*boxx);
    xyzq_val.y = (float)(y[tid] + ((double)shift.y)*boxy);
    xyzq_val.z = (float)(z[tid] + ((double)shift.z)*boxz);
    xyzq_val.w = q[loc2glo[tid]];
    xyzq[tid] = xyzq_val;
  }
}

//
// Copies (x, y, z) into xyzq -array and also shifts (x, y, z)
//
__global__ void set_xyz_shift_kernel(const int ncoord,
				     const double* __restrict__ x,
				     const double* __restrict__ y,
				     const double* __restrict__ z,
				     const float3* __restrict__ xyz_shift,
				     const double boxx, const double boxy, const double boxz,
				     float4* __restrict__ xyzq) {
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if (tid < ncoord) {
    float4 xyzq_val;
    float3 shift = xyz_shift[tid];
    xyzq_val.x = (float)(x[tid] + ((double)shift.x)*boxx);
    xyzq_val.y = (float)(y[tid] + ((double)shift.y)*boxy);
    xyzq_val.z = (float)(z[tid] + ((double)shift.z)*boxz);
    xyzq[tid].x = xyzq_val.x;
    xyzq[tid].y = xyzq_val.y;
    xyzq[tid].z = xyzq_val.z;
  }
}

//##########################################################################################
//##########################################################################################
//##########################################################################################

//
// Return xyzq length that has extra align:
// ncoord-1 = last possible index
//
int XYZQ::get_xyzq_len(const int ncoord_in) {
  return ((ncoord_in-1)/align+1)*align;
}

//
// Class creator
//
XYZQ::XYZQ() {
  ncoord = 0;
  xyzq_len = 0;
  align = warpsize;
  xyzq = NULL;
}

//
// Class creator
//
XYZQ::XYZQ(int ncoord, int align) : ncoord(ncoord), align(align) {
  xyzq_len = get_xyzq_len(ncoord);
  allocate<float4>(&xyzq, xyzq_len);
}

//
// Class creator
//
XYZQ::XYZQ(const char *filename, int align) : align(align) {
  
  std::ifstream file(filename);
  if (file.is_open()) {
    
    float x, y, z, q;
    
    // Count number of coordinates
    ncoord = 0;
    while (file >> x >> y >> z >> q) ncoord++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    float4 *xyzq_cpu = new float4[ncoord];
    
    // Read coordinates
    int i=0;
    while (file >> xyzq_cpu[i].x >> xyzq_cpu[i].y >> xyzq_cpu[i].z >> xyzq_cpu[i].w) i++;
    
    // Allocate GPU memory
    xyzq_len = get_xyzq_len(ncoord);
    allocate<float4>(&xyzq, xyzq_len);

    // Copy coordinates from CPU to GPU
    copy_HtoD<float4>(xyzq_cpu, xyzq, ncoord);

    // Deallocate CPU memory
    delete [] xyzq_cpu;
    
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }
  
}

//
// Class destructor
//
XYZQ::~XYZQ() {
  if (xyzq != NULL) deallocate<float4>(&xyzq);
}

//
// Re-allocates array, does not preserve content
//
void XYZQ::realloc(int ncoord_new, float fac) {
  reallocate<float4>(&xyzq, &xyzq_len, get_xyzq_len(ncoord_new), fac);
  this->ncoord = ncoord_new;
}

//
// Re-sizes array, preserves content
//
void XYZQ::resize(int ncoord_new, float fac) {
  ::resize<float4>(&xyzq, &xyzq_len, ncoord, get_xyzq_len(ncoord_new), fac);
  this->ncoord = ncoord_new;
}

//
// Copies xyzq from host
// NOTE: Does not reallocate xyzq
//
void XYZQ::set_xyzq(int ncopy, float4 *h_xyzq, size_t offset, hipStream_t stream) {
  copy_HtoD<float4>(&h_xyzq[offset], &xyzq[offset], ncopy, stream);
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double>& coord, const float *q, hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyzq_kernel<<< nblock, nthread, 0, stream >>>
    (coord.size(), coord.x(), coord.y(), coord.z(), q, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyzq(const cudaXYZ<double>& coord, const float *q, const int *loc2glo,
		    const float3 *xyz_shift,
		    const double boxx, const double boxy, const double boxz, hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyzq_shift_kernel<<< nblock, nthread, 0, stream >>>
    (coord.size(), coord.x(), coord.y(), coord.z(), q,
     loc2glo, xyz_shift, boxx, boxy, boxz, xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double>& coord, hipStream_t stream) {
  assert(ncoord == coord.size());

  int nthread = 512;
  int nblock = (ncoord-1)/nthread+1;

  set_xyz_kernel<<< nblock, nthread, 0, stream >>>
    (coord.size(), coord.x(), coord.y(), coord.z(), xyzq);

  cudaCheck(hipGetLastError());
}

//
// Copies x,y,z,q (on device) into the coordinate slots
//
void XYZQ::set_xyz(const cudaXYZ<double>& coord, const int start, const int end, const float3 *xyz_shift,
		   const double boxx, const double boxy, const double boxz, hipStream_t stream) {
  assert(ncoord == coord.size());
  assert(start >= 0);
  assert(end < ncoord);
  assert(end < coord.size());
  int nset = end-start+1;
  assert(nset >= 0);

  if (nset == 0) return;

  int nthread = 512;
  int nblock = (nset-1)/nthread+1;

  set_xyz_shift_kernel<<< nblock, nthread, 0, stream >>>
    (nset, coord.x()+start, coord.y()+start, coord.z()+start,
     &xyz_shift[start], boxx, boxy, boxz, &xyzq[start]);

  cudaCheck(hipGetLastError());
}

//
// Compares two XYZQ arrays
//
bool XYZQ::compare(XYZQ& xyzq_in, const double tol, double& max_diff) {
  assert(xyzq_in.ncoord == ncoord);

  float4 *h_xyzq = new float4[ncoord];
  float4 *h_xyzq_in = new float4[ncoord];
  copy_DtoH<float4>(xyzq, h_xyzq, ncoord);
  copy_DtoH<float4>(xyzq_in.xyzq, h_xyzq_in, ncoord);

  bool ok = true;

  max_diff = 0.0;
  int i;
  double dx, dy, dz, dq;
  double diff;
  try {
    for (i=0;i < ncoord;i++) {
      dx = fabs(h_xyzq[i].x - h_xyzq_in[i].x);
      dy = fabs(h_xyzq[i].y - h_xyzq_in[i].y);
      dz = fabs(h_xyzq[i].z - h_xyzq_in[i].z);
      dq = fabs(h_xyzq[i].w - h_xyzq_in[i].w);
      diff = max(dx, max(dy, dz));
      max_diff = max(max_diff, diff);
      if (diff > tol || dq > 0.0) throw 1;
    }
  }
  catch (int a) {
    std::cout << "i = " << i << std::endl;
    std::cout << "this: x,y,z,q = " << h_xyzq[i].x << " " << h_xyzq[i].y
	      << " " << h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
    std::cout << "in  : x,y,z,q = " << h_xyzq_in[i].x << " " << h_xyzq_in[i].y
	      << " " << h_xyzq_in[i].z << " " << h_xyzq_in[i].w << std::endl;
    ok = false;
  }

  delete [] h_xyzq;
  delete [] h_xyzq_in;

  return ok;
}

//
// Print to ostream
//
void XYZQ::print(const int start, const int end, std::ostream& out) {

  float4 *h_xyzq = new float4[ncoord];
  copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);

  for (int i=start;i <= end;i++) {
    out << i << " " << h_xyzq[i].x << " " << h_xyzq[i].y << " "
	<< h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
  }

  delete [] h_xyzq;
}

//
// Save to file
//
void XYZQ::save(const char* filename) {
  std::ofstream file(filename);
  if (file.is_open()) {
    float4 *h_xyzq = new float4[ncoord];
    copy_DtoH_sync<float4>(xyzq, h_xyzq, ncoord);
    for (int i=0;i < ncoord;i++) {
      file << h_xyzq[i].x << " " << h_xyzq[i].y << " "
	   << h_xyzq[i].z << " " << h_xyzq[i].w << std::endl;
    }
    delete [] h_xyzq;
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }
}
