#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include "gpu_utils.h"
#include "reduce.h"
#include "cuda_utils.h"
#include "Force.h"
#include "hostXYZ.h"

template <typename T>
Force<T>::Force(const char *filename) {
  _size = 0;
  _stride = 0;
  _capacity = 0;
  _xyz = NULL;

  std::ifstream file(filename);
  if (file.is_open()) {
    
    T fx, fy, fz;
    
    // Count number of coordinates
    int nforce = 0;
    while (file >> fx >> fy >> fz) nforce++;

    // Rewind
    file.clear();
    file.seekg(0, std::ios::beg);
    
    // Allocate CPU memory
    hostXYZ<T> xyz_cpu(nforce, NON_PINNED);

    // Read coordinates
    int i=0;
    while (file >> xyz_cpu.x()[i] >> xyz_cpu.y()[i] >> xyz_cpu.z()[i]) i++;

    // Allocate GPU memory
    this->realloc(nforce);

    // Copy coordinates from CPU to GPU
    copy_HtoD_sync<T>(xyz_cpu.x(), this->x(), nforce);
    copy_HtoD_sync<T>(xyz_cpu.y(), this->y(), nforce);
    copy_HtoD_sync<T>(xyz_cpu.z(), this->z(), nforce);

  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }

}

//
// Compares two force arrays, returns true if the difference is within tolerance
// NOTE: Comparison is done in double precision
//
template <typename T>
bool Force<T>::compare(Force<T>& force, const double tol, double& max_diff) {
  assert(force.size() == this->size());

  hostXYZ<T> xyz1(this->size(), NON_PINNED);
  hostXYZ<T> xyz2(force.size(), NON_PINNED);
  xyz1.set_data_sync(force.size(), force.x(), force.y(), force.z());
  xyz2.set_data_sync(this->size(), this->x(), this->y(), this->z());

  max_diff = 0.0;

  int i;
  double fx1, fy1, fz1;
  double fx2, fy2, fz2;
  double diff;
  for (i=0;i < this->size();i++) {
    fx1 = (double)(xyz1.x()[i]);
    fy1 = (double)(xyz1.y()[i]);
    fz1 = (double)(xyz1.z()[i]);
    fx2 = (double)(xyz2.x()[i]);
    fy2 = (double)(xyz2.y()[i]);
    fz2 = (double)(xyz2.z()[i]);
    if (isnan(fx1) || isnan(fy1) || isnan(fz1) || isnan(fx2) || isnan(fy2) || isnan(fz2)) {
      std::cout << "i = "<< i << std::endl;
      std::cout << "this: fx1 fy1 fz1 = " << fx1 << " "<< fy1 << " "<< fz1 << std::endl;
      std::cout << "force:fx2 fy2 fz2 = " << fx2 << " "<< fy2 << " "<< fz2 << std::endl;
      return false;
    }
    diff = max(fabs(fx1-fx2), max(fabs(fy1-fy2), fabs(fz1-fz2)));
    max_diff = max(diff, max_diff);
    if (diff > tol) {
      std::cout << "i = "<< i << std::endl;
      std::cout << "this: fx1 fy1 fz1 = " << fx1 << " "<< fy1 << " "<< fz1 << std::endl;
      std::cout << "force:fx2 fy2 fz2 = " << fx2 << " "<< fy2 << " "<< fz2 << std::endl;
      std::cout << "difference: " << diff << std::endl;
      return false;
    }
  }

  return true;
}

//
// Converts one type of force array to another. Result is in "force"
//
template <typename T>
template <typename T2>
void Force<T>::convert(Force<T2>& force, hipStream_t stream) {

  assert(force.size() == this->size());

  if (force.stride() == this->stride()) {
    int nthread = 512;
    int nblock = (3*this->stride() - 1)/nthread + 1;
    reduce_force<T, T2>
      <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz(), force.xyz());
    cudaCheck(hipGetLastError());
  } else {
    int nthread = 512;
    int nblock = (this->size() - 1)/nthread + 1;
    reduce_force<T, T2>
      <<< nblock, nthread, 0, stream >>>(this->size(), this->stride(), this->xyz(),
					 force.stride(), force.xyz());
    cudaCheck(hipGetLastError());
  }
}

//
// Converts one type of force array to another. Result is in "force"
//
template <typename T>
template <typename T2, typename T3>
void Force<T>::convert_to(Force<T3>& force, hipStream_t stream) {

  assert(force.size() == this->size());
  assert(force.stride() == this->stride());
  assert(sizeof(T2) == sizeof(T3));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_force<T, T2>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz(), (T2 *)force.xyz());
  cudaCheck(hipGetLastError());
}

//
// Converts one type of force array to another. Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2>
void Force<T>::convert(hipStream_t stream) {

  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_force<T, T2>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Converts one type of force array to another and adds force to the result.
// Result is in "this"
// NOTE: Only works when the size of the types T and T2 match
//
template <typename T>
template <typename T2, typename T3>
void Force<T>::convert_add(Force<T3>& force, hipStream_t stream) {
  assert(force.stride() == this->stride());
  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (3*this->stride() - 1)/nthread + 1;

  reduce_add_force<T, T2, T3>
    <<< nblock, nthread, 0, stream >>>(3*this->stride(), force.xyz(), this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Adds non-strided force_data
//
template <typename T>
template <typename T2>
void Force<T>::add(float3 *force_data, int force_n, hipStream_t stream) {

  assert(force_n <= this->size());
  assert(sizeof(T) == sizeof(T2));

  int nthread = 512;
  int nblock = (force_n - 1)/nthread + 1;

  add_nonstrided_force<<< nblock, nthread, 0, stream >>>
    (force_n, force_data, this->stride(), (double *)this->xyz());
  cudaCheck(hipGetLastError());
}

//
// Save to file
//
template <typename T>
template <typename T2>
void Force<T>::save(const char* filename) {
  assert(sizeof(T) == sizeof(T2));
  std::ofstream file(filename);
  if (file.is_open()) {
    T2 *h_xyz = new T2[_stride*3];
    copy_DtoH_sync<T2>((T2 *)_xyz, h_xyz, _stride*3);
    for (int i=0;i < _size;i++) {
      file << h_xyz[i] << " "
	   << h_xyz[i + _stride] << " "
	   << h_xyz[i + _stride*2] << std::endl;
    }
    delete [] h_xyz;
  } else {
    std::cerr<<"Error opening file "<<filename<<std::endl;
    exit(1);
  }
}

//
// Gets forces to host
//
template <typename T>
void Force<T>::getXYZ(T* h_x, T* h_y, T* h_z) {
  T *h_xyz = new T[_stride*3];
  copy_DtoH_sync<T>(_xyz, h_xyz, _stride*3);
  for (int i=0;i < _size;i++) {
    h_x[i] = h_xyz[i];
    h_y[i] = h_xyz[i + _stride];
    h_z[i] = h_xyz[i + _stride*2];
  }
  delete [] h_xyz;
}

//
// Explicit instances of Force class
//
template class Force<long long int>;
template class Force<double>;
template class Force<float>;
template void Force<long long int>::convert<double>(hipStream_t stream);
template void Force<long long int>::convert_add<double>(Force<float>& force, hipStream_t stream);
template void Force<long long int>::convert<float>(Force<float>& force, hipStream_t stream);
template void Force<long long int>::convert<double>(Force<double>& force, hipStream_t stream);
template void Force<float>::convert_to<double>(Force<long long int>& force, hipStream_t stream);
template void Force<long long int>::add<double>(float3 *force_data, int force_n, hipStream_t stream);
template void Force<long long int>::save<double>(const char* filename);
template void Force<double>::save<double>(const char* filename);
template void Force<float>::save<float>(const char* filename);
