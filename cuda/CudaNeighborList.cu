#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cassert>
#include <fstream>
#include <vector>
#include <algorithm>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "CudaNeighborList.h"

// IF defined, uses strict (Factor = 1.0f) memory reallocation. Used for debuggin memory problems.
#define STRICT_MEMORY_REALLOC

//static const int numNlistParam=2;
//static __device__ NeighborListParam_t d_NlistParam[numNlistParam];

//static __device__ ZoneParam_t d_ZoneParam[maxNumZone];

//########################################################################################
//########################################################################################
//########################################################################################

//
// Class creator
//
template <int tilesize>
CudaNeighborList<tilesize>::CudaNeighborList(const CudaTopExcl& topExcl,
					     const int nx, const int ny, const int nz) : 
  topExcl(topExcl) {

  numZone = 0;

  numList = 0;

  ind_sorted_len = 0;
  ind_sorted = NULL;

  col_ncellz_len = 0;
  col_ncellz = NULL;

  col_cell_len = 0;
  col_cell = NULL;

  cell_patom_len = 0;
  cell_patom = NULL;

  cell_xyz_zone_len = 0;
  cell_xyz_zone = NULL;

  cell_bz_len = 0;
  cell_bz = NULL;

  bb_len = 0;
  bb = NULL;

  d_ZoneParam_len = 0;
  d_ZoneParam = NULL;

  h_ZoneParam_len = 0;
  h_ZoneParam = NULL;

  imx_lo = 0;
  imx_hi = 0;
  imy_lo = 0;
  imy_hi = 0;
  imz_lo = 0;
  imz_hi = 0;
  if (nx == 1) {
    imx_lo = -1;
    imx_hi = 1;
  }
  if (ny == 1) {
    imy_lo = -1;
    imy_hi = 1;
  }
  if (nz == 1) {
    imz_lo = -1;
    imz_hi = 1;
  }

  cudaCheck(hipEventCreate(&glo2loc_reset_event));

  test = false;
}

//
// Class destructor
//
template <int tilesize>
CudaNeighborList<tilesize>::~CudaNeighborList() {
  // Neighbor list building
  if (ind_sorted != NULL) deallocate<int>(&ind_sorted);
  if (cell_patom != NULL) deallocate<int>(&cell_patom);
  if (col_ncellz != NULL) deallocate<int>(&col_ncellz);
  if (col_cell != NULL) deallocate<int>(&col_cell);
  if (cell_xyz_zone != NULL) deallocate<int4>(&cell_xyz_zone);
  if (cell_bz != NULL) deallocate<float>(&cell_bz);
  if (bb != NULL) deallocate<bb_t>(&bb);
  for (int i=0;i < d_NlistParam.size();i++) {
    deallocate<NlistParam_t>(&d_NlistParam.at(i));
    deallocate_host<NlistParam_t>(&h_NlistParam.at(i));
  }
  for (int i=0;i < sorter.size();i++) {
    delete sorter.at(i);
    delete builder.at(i);
  }
  if (h_ZoneParam != NULL) deallocate_host<ZoneParam_t>(&h_ZoneParam);
  if (d_ZoneParam != NULL) deallocate<ZoneParam_t>(&d_ZoneParam);
  for (int i=0;i < build_event.size();i++) {
    cudaCheck(hipEventDestroy(build_event.at(i)));
  }
  cudaCheck(hipEventDestroy(glo2loc_reset_event));
}

//
// Register List
//
template <int tilesize>
void CudaNeighborList<tilesize>::registerList(std::vector<int>& numIntZone,
					      std::vector< std::vector<int> >& intZones,
					      const char *filename) {
  assert(numIntZone.size() == intZones.size());
  assert(numIntZone.size() <= maxNumZone);

  numList++;

  // Get izoneStart and izoneEnd
  int izoneStart = numZone+1;
  int izoneEnd = -1;
  for (int izone=0;izone < numIntZone.size();izone++) {
    if (numIntZone.at(izone) > 0) {
      izoneStart = min(izoneStart, izone);
      izoneEnd   = max(izoneEnd, izone);
    }
  }
  numZone = max(numZone, izoneEnd+1);

  resize_host<ZoneParam_t>(&h_ZoneParam, &h_ZoneParam_len, h_ZoneParam_len, numZone, 1.0f);
  resize<ZoneParam_t>(&d_ZoneParam, &d_ZoneParam_len, d_ZoneParam_len, numZone, 1.0f);

  // ----------------------
  // Setup h_ZoneParam
  // ----------------------
  int n_int_zone_max = 0;
  for (int izone=izoneStart;izone <= izoneEnd;izone++) {
    h_ZoneParam[izone].n_int_zone = numIntZone.at(izone);
    if (h_ZoneParam[izone].n_int_zone > 0) {
      assert(intZones.at(izone).size() <= maxNumIntZone);
      std::copy(intZones.at(izone).begin(), intZones.at(izone).end(), h_ZoneParam[izone].int_zone);
      n_int_zone_max = max(n_int_zone_max, h_ZoneParam[izone].n_int_zone);
    }
  }  

  // Create list sorter and builder
  sorter.push_back(new CudaNeighborListSort(tilesize, izoneStart, izoneEnd));
  if (filename != NULL) {
    builder.push_back(new CudaNeighborListBuild<tilesize>(n_int_zone_max, izoneStart, izoneEnd, filename));
  } else {
    builder.push_back(new CudaNeighborListBuild<tilesize>(n_int_zone_max, izoneStart, izoneEnd));
  }

  // Create new NlistParam
  NlistParam_t *d_tmp;
  allocate<NlistParam_t>(&d_tmp, 1);
  NlistParam_t *h_tmp;
  allocate_host<NlistParam_t>(&h_tmp, 1);
  d_NlistParam.push_back(d_tmp);
  h_NlistParam.push_back(h_tmp);

  int indList = numList-1;

  h_NlistParam.at(indList)->imx_lo = imx_lo;
  h_NlistParam.at(indList)->imx_hi = imx_hi;
  h_NlistParam.at(indList)->imy_lo = imy_lo;
  h_NlistParam.at(indList)->imy_hi = imy_hi;
  h_NlistParam.at(indList)->imz_lo = imz_lo;
  h_NlistParam.at(indList)->imz_hi = imz_hi;

  copy_HtoD_sync<NlistParam_t>(h_NlistParam.at(indList), d_NlistParam.at(indList), 1);

  // Create events
  build_event.resize(numList);
  cudaCheck(hipEventCreate(&build_event.at(indList)));
}

//
// Sort
//
template <int tilesize>
void CudaNeighborList<tilesize>::sort(const int indList, const int *zone_patom,
				      const float4 *xyzq,
				      float4 *xyzq_sorted,
				      int *loc2glo,
				      hipStream_t stream) {
  assert(indList >= 0 && indList < numList);

  // Reset glo2loc or wait for the reset to be done
  if (indList == 0) {
    set_gpu_array<int>(topExcl.get_glo2loc(), topExcl.get_ncoord(), -1, stream);
    cudaCheck(hipEventRecord(glo2loc_reset_event, stream));
  } else {
    cudaCheck(hipStreamWaitEvent(stream, glo2loc_reset_event, 0));
  }

  // ------------------------ min/max -----------------------------
  sorter.at(indList)->calc_min_max_xyz(zone_patom, xyzq,
				      h_ZoneParam, d_ZoneParam, stream);
  // --------------------------------------------------------------

  // -------------------------- Setup -----------------------------
  // NOTE: After this call, ncol_tot, ncoord_tot, and ncell_max are up to date
  sorter.at(indList)->sort_setup(zone_patom, h_ZoneParam, d_ZoneParam, stream);
  // --------------------------------------------------------------

  // ------------------------ Realloc -----------------------------
  sort_realloc(indList);
  // --------------------------------------------------------------

  // ---------------------- Do actual sorting ---------------------
  int cellStart = 0;
  int colStart = 0;
  for (int i=0;i < indList;i++) {
    cellStart += sorter.at(i)->get_ncell();
    colStart += sorter.at(i)->get_ncol_tot();
  }
  sorter.at(indList)->sort_core(zone_patom, cellStart, colStart,
				h_ZoneParam, d_ZoneParam, d_NlistParam.at(indList),
				cell_patom, col_ncellz, cell_xyz_zone,
				col_cell, ind_sorted, xyzq, xyzq_sorted, stream);
  // --------------------------------------------------------------

  // ------------------ Build indices etc. after sort -------------
  // NOTE: After this call, ncell is up to date
  sorter.at(indList)->sort_build_indices(zone_patom, cellStart, cell_patom,
					 xyzq_sorted, loc2glo, topExcl.get_glo2loc(),
					 ind_sorted, bb, cell_bz, stream);
  // --------------------------------------------------------------

  // Test sort
  if (test) {
    sorter.at(indList)->test_sort(zone_patom, cellStart, h_ZoneParam, xyzq, xyzq_sorted,
				  ind_sorted, cell_patom);
  }
}

//
// Allocates / Re-allocates memory for sort
//
template <int tilesize>
void CudaNeighborList<tilesize>::sort_realloc(const int indList) {

  sorter.at(indList)->sort_realloc();

#ifdef STRICT_MEMORY_REALLOC
  float fac = 1.0f;
#else
  float fac = 1.2f;
#endif

  // Compute ncol_tot, ncoord_tot, ncell_max.
  // NOTE: these variables are quaranteed to be up to date because
  //       sort is only launched after launching the previous build
  // Current content
  int ncol_cur = 0;
  int ncoord_cur = 0;
  int ncell_cur = 0;
  for (int i=0;i < indList;i++) {
    ncol_cur += sorter.at(i)->get_ncol_tot();
    ncoord_cur += sorter.at(i)->get_ncoord_tot();
    // NOTE: here we're using ncell instead of ncell_max, because this value
    //       from previous launch is now up to date
    ncell_cur += sorter.at(i)->get_ncell();
  }
  // Total
  int ncol_tot = ncol_cur + sorter.at(indList)->get_ncol_tot();
  int ncoord_tot = ncoord_cur + sorter.at(indList)->get_ncoord_tot();
  int ncell_tot = ncell_cur + sorter.at(indList)->get_ncell_max();

  bool q_resize = (cell_patom_len < ncell_tot+1) || (cell_xyz_zone_len < ncell_tot) ||
    (cell_bz_len < ncell_tot) || (bb_len < ncell_tot) || (col_ncellz_len < ncol_tot) ||
    (col_cell_len < ncol_tot) || (ind_sorted_len < ncoord_tot);

  if (q_resize) {
    // We must wait here until all preceding neighbor list builds have finished before
    // reallocating these arrays.
    for (int i=0;i < indList;i++) {
      cudaCheck(hipEventSynchronize(build_event.at(i)));
    }

    resize<int>(&cell_patom, &cell_patom_len, ncell_cur+1, ncell_tot+1, fac);
    resize<int4>(&cell_xyz_zone, &cell_xyz_zone_len, ncell_cur, ncell_tot, fac);
    resize<float>(&cell_bz, &cell_bz_len, ncell_cur, ncell_tot, fac);
    resize<bb_t>(&bb, &bb_len, ncell_cur, ncell_tot, fac);

    resize<int>(&col_ncellz, &col_ncellz_len, ncol_cur, ncol_tot, fac);
    resize<int>(&col_cell, &col_cell_len, ncol_cur, ncol_tot, fac);

    resize<int>(&ind_sorted, &ind_sorted_len, ncoord_cur, ncoord_tot, fac);
  }

}


//
// Build
//
template <int tilesize>
void CudaNeighborList<tilesize>::build(const int indList, const int* zone_patom,
				       const float boxx, const float boxy, const float boxz,
				       const float rcut,
				       const float4 *xyzq, const int *loc2glo,
				       hipStream_t stream) {
  assert(indList >= 0 && indList < numList);

  int cellStart = 0;
  for (int i=0;i < indList;i++) {
    cellStart += sorter.at(i)->get_ncell();
  }

  builder.at(indList)->build(sorter.at(indList)->get_ncell(), cellStart, topExcl.getMaxNumExcl(),
			     h_ZoneParam, d_ZoneParam,
			     boxx, boxy, boxz, rcut, xyzq, loc2glo, topExcl.get_glo2loc(),
			     topExcl.getAtomExclPos(), topExcl.getAtomExcl(),
			     cell_xyz_zone, col_ncellz, col_cell, cell_bz, cell_patom, bb, 
			     h_NlistParam.at(indList), d_NlistParam.at(indList), stream);
  cudaCheck(hipEventRecord(build_event.at(indList), stream));

  /*
  // Check for blown arrays
  copy_DtoH<NlistParam_t>(d_NlistParam.at(indList), h_NlistParam.at(indList), 1, stream);
  cudaCheck(hipStreamSynchronize(stream));

  int n_ientry = h_NlistParam.at(indList)->n_ientry;
  int n_tile = h_NlistParam.at(indList)->n_tile;

  if (n_tile > builder.at(indList)->get_n_tile_est()) {
    std::cout << "CudaNeighborListBuild::build, Limit blown: n_tile > n_tile_est"<< std::endl;
    exit(1);
  }

  if (n_ientry > builder.at(indList)->get_n_ientry_est()) {
    std::cout << "CudaNeighborListBuild::build, Limit blown: n_ientry > n_ientry_est"<< std::endl;
    exit(1);
  }
  */

  if (test) {
    int ncol_tot = 0;
    int ncell_tot = 0;
    for (int i=0;i <= indList;i++) {
      ncol_tot += sorter.at(i)->get_ncol_tot();
      ncell_tot += sorter.at(i)->get_ncell();
    }
    builder.at(indList)->test_build(zone_patom, ncol_tot, ncell_tot, h_ZoneParam,
				    topExcl.getAtomExclPosLen(), topExcl.getAtomExclPos(),
				    topExcl.getAtomExclLen(), topExcl.getAtomExcl(), 
				    boxx, boxy, boxz, rcut, xyzq, loc2glo, topExcl.get_glo2loc(),
				    topExcl.get_ncoord(),
				    cell_patom, col_cell, cell_bz, bb);
    builder.at(indList)->analyze();
  }
}

/*
//
// Copies h_NlistParam (CPU) -> d_NlistParam (GPU)
//
template <int tilesize>
void CudaNeighborList<tilesize>::set_NlistParam(hipStream_t stream) {
  cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_NlistParam), h_NlistParam, sizeof(NeighborListParam_t),
  				    0, hipMemcpyHostToDevice, stream));
}

//
// Copies d_NlistParam (GPU) -> h_NlistParam (CPU)
//
template <int tilesize>
void CudaNeighborList<tilesize>::NlistParam(hipStream_t stream) {
  //cudaCheck(hipMemcpyFromSymbol(h_NlistParam, HIP_SYMBOL(d_NlistParam), sizeof(NeighborListParam_t),
  //				 0, hipMemcpyDeviceToHost));
}
*/

/*
//
// Copies h_ZoneParam (CPU) -> d_ZoneParam (GPU)
//
template <int tilesize>
void CudaNeighborList<tilesize>::setZoneParam(hipStream_t stream) {
  cudaCheck(hipMemcpyToSymbolAsync(HIP_SYMBOL(&d_ZoneParam[izoneStart]), h_ZoneParam, sizeof(ZoneParam_t),
  				    0, hipMemcpyHostToDevice, stream));
}

//
// Copies d_ZoneParam (GPU) -> h_ZoneParam (CPU)
//
template <int tilesize>
void CudaNeighborList<tilesize>::getZoneParam() {
  cudaCheck(hipMemcpyFromSymbol(h_ZoneParam, HIP_SYMBOL(d_ZoneParam), sizeof(ZoneParam_t),
				 0, hipMemcpyDeviceToHost));
}

//
// Resets n_tile and n_ientry variables for build() -call
//
template <int tilesize>
void CudaNeighborList<tilesize>::reset() {
  get_NlistParam();
  cudaCheck(hipDeviceSynchronize());
  h_NlistParam->n_tile = 0;
  h_NlistParam->n_ientry = 0;
  set_NlistParam(0);
  cudaCheck(hipDeviceSynchronize());
}
*/

//
// Explicit instances of CudaNeighborList
//
template class CudaNeighborList<32>;
