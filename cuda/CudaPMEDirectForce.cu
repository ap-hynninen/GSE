#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cassert>
#include <hip/hip_runtime.h>
#include <math.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "CudaPMEDirectForce.h"
#include "CudaDirectForceKernels.h"

//
// Sets vdwtype from a global list
//
__global__ void set_vdwtype_kernel(const int ncoord, const int* __restrict__ glo_vdwtype,
				   const int* __restrict__ loc2glo, int* __restrict__ vdwtype) {
  const int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < ncoord) {
    int j = loc2glo[i];
    vdwtype[i] = glo_vdwtype[j];
  }
}

__global__ void set_14_list_kernel(const int nin14_tbl, const int* __restrict__ in14_tbl,
				   const xx14_t* __restrict__ in14, xx14list_t* __restrict__ in14list,
				   const int nex14_tbl, const int* __restrict__ ex14_tbl,
				   const xx14_t* __restrict__ ex14, xx14list_t* __restrict__ ex14list,
				   const float4* __restrict__ xyzq,
				   const float3 half_box, const int*__restrict__ glo2loc_ind) {
  int pos = threadIdx.x + blockIdx.x*blockDim.x;
  if (pos < nin14_tbl) {
    int j = in14_tbl[pos];
    xx14_t in14v = in14[j];
    xx14list_t in14listv;
    in14listv.i = glo2loc_ind[in14v.i];
    in14listv.j = glo2loc_ind[in14v.j];
    float4 xyzq_i = xyzq[in14listv.i];
    float4 xyzq_j = xyzq[in14listv.j];
    in14listv.ishift = calc_ishift(xyzq_i, xyzq_j, half_box);
    in14list[pos] = in14listv;
  } else if (pos < nin14_tbl + nex14_tbl) {
    pos -= nin14_tbl;
    int j = ex14_tbl[pos];
    xx14_t ex14v = ex14[j];
    xx14list_t ex14listv;
    ex14listv.i = glo2loc_ind[ex14v.i];
    ex14listv.j = glo2loc_ind[ex14v.j];
    float4 xyzq_i = xyzq[ex14listv.i];
    float4 xyzq_j = xyzq[ex14listv.j];
    ex14listv.ishift = calc_ishift(xyzq_i, xyzq_j, half_box);
    ex14list[pos] = ex14listv;
  }
}

//########################################################################################
//########################################################################################
//########################################################################################

//
// Class creator
//
template <typename AT, typename CT>
CudaPMEDirectForce<AT, CT>::CudaPMEDirectForce(CudaEnergyVirial &energyVirial,
					       const char *nameVdw, const char *nameElec, const char *nameExcl) : 
  use_tex_vdwparam(true), use_tex_vdwparam14(true), energyVirial(energyVirial) {

  assert(nameVdw != NULL);
  assert(nameElec != NULL);
  assert(nameExcl != NULL);
  
  // Insert energy terms
  energyVirial.insert(nameVdw);
  strVdw = nameVdw;

  energyVirial.insert(nameElec);
  strElec = nameElec;

  energyVirial.insert(nameExcl);
  strExcl = nameExcl;
  
#ifdef USE_TEXTURE_OBJECTS
  vdwParamTexObjActive = false;
  vdwParam14TexObjActive = false;
#else
  // Assert that texture references must be unbound
  assert(!get_vdwparam_texref_bound());
  assert(!get_vdwparam14_texref_bound());
#endif 

  vdwparam = NULL;
  nvdwparam = 0;
  vdwparam_len = 0;

  vdwparam14 = NULL;
  nvdwparam14 = 0;
  vdwparam14_len = 0;

  nin14list = 0;
  in14list_len = 0;
  in14list = NULL;

  nex14list = 0;
  ex14list_len = 0;
  ex14list = NULL;

  vdwtype = NULL;
  vdwtype_len = 0;

  ewald_force = NULL;
  n_ewald_force = 0;

  set_calc_vdw(true);
  set_calc_elec(true);

  //allocate<DirectEnergyVirial_t>(&d_energy_virial, 1);
  //allocate_host<DirectEnergyVirial_t>(&h_energy_virial, 1);
  
  allocate_host<DirectSettings_t>(&h_setup, 1);

  //clear_energy_virial();
}

//
// Class destructor
//
template <typename AT, typename CT>
CudaPMEDirectForce<AT, CT>::~CudaPMEDirectForce() {
  this->clearTextures();
  if (vdwparam != NULL) deallocate<CT>(&vdwparam);
  if (vdwparam14 != NULL) deallocate<CT>(&vdwparam14);
  if (in14list != NULL) deallocate<xx14list_t>(&in14list);
  if (ex14list != NULL) deallocate<xx14list_t>(&ex14list);
  if (vdwtype != NULL) deallocate<int>(&vdwtype);
  if (ewald_force != NULL) deallocate<CT>(&ewald_force);
  //if (d_energy_virial != NULL) deallocate<DirectEnergyVirial_t>(&d_energy_virial);
  //if (h_energy_virial != NULL) deallocate_host<DirectEnergyVirial_t>(&h_energy_virial);
  if (h_setup != NULL) deallocate_host<DirectSettings_t>(&h_setup);
}

//
// Copies h_setup -> d_setup
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::update_setup() {
  updateDirectForceSetup(h_setup);
}

//
// Sets parameters for the nonbonded computation
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::setup(double boxx, double boxy, double boxz, double kappa,
				       double roff, double ron, double e14fac,
				       int vdw_model, int elec_model) {

  double ron2 = ron*ron;
  double ron4 = ron2*ron2;
  double ron3 = ron*ron*ron;
  double ron5 = ron3*ron2;
  double ron6 = ron3*ron3;

  double roff2 = roff*roff;
  double roff3 = roff*roff*roff;
  double roff4 = roff2*roff2;
  double roff5 = roff4*roff;
  double roff6 = roff3*roff3;
  double roff8 = roff6*roff2;
  double roff12 = roff6*roff6;
  double roff14 = roff12*roff2;

  h_setup->boxx = boxx;
  h_setup->boxy = boxy;
  h_setup->boxz = boxz;
  h_setup->kappa = kappa;
  h_setup->kappa2 = kappa*kappa;
  h_setup->roff = roff;
  h_setup->roff2 = roff2;
  h_setup->roff3 = roff3;
  h_setup->roff5 = roff5;
  h_setup->ron = ron;
  h_setup->ron2 = ron2;

  h_setup->roffinv =  ((CT)1.0)/roff;
  h_setup->roffinv2 =  ((CT)1.0)/roff2;
  h_setup->roffinv3 =  ((CT)1.0)/roff3;
  h_setup->roffinv4 = ((CT)1.0)/(roff2*roff2);
  h_setup->roffinv5 = ((CT)1.0)/(roff*roff2*roff2);
  h_setup->roffinv6 = ((CT)1.0)/(roff2*roff2*roff2);
  h_setup->roffinv12 = h_setup->roffinv6*h_setup->roffinv6;
  h_setup->roffinv18 = h_setup->roffinv12*h_setup->roffinv6;

  double roff2_min_ron2 = roff2 - ron2;
  double inv_roff2_ron2_3 = 1.0/(roff2_min_ron2*roff2_min_ron2*roff2_min_ron2);
  h_setup->inv_roff2_ron2_3 = (CT)inv_roff2_ron2_3;
 
  // Constants for VFSW
  if (ron < roff) {
    h_setup->k6 = roff3/(roff3 - ron3);
    h_setup->k12 = roff6/(roff6 - ron6);
    h_setup->dv6 = -((CT)1.0)/(ron3*roff3);
    h_setup->dv12 = -((CT)1.0)/(ron6*roff6);
  } else {
    h_setup->k6 = 1.0f;
    h_setup->k12 = 1.0f;
    h_setup->dv6 = -((CT)1.0)/(roff6);
    h_setup->dv12 = -((CT)1.0)/(roff6*roff6);
  }

  double g = (roff2 - ron2)*(roff2 - ron2)*(roff2 - ron2);
  h_setup->Aconst = roff4*(roff2 - 3.0*ron2)/g;
  h_setup->Bconst = 6.0*roff2*ron2/g;
  h_setup->Cconst = -(ron2 + roff2)/g;
  h_setup->Dconst = 2.0/(5.0*g);
  h_setup->dvc = 8.0*(ron2*roff2*(roff-ron) - (roff5 - ron5)/5.0)/g;
  if (ron < roff) {
    double Denom = 1.0/g;
    h_setup->Denom = Denom;
    double Acoef = roff4*(roff2 - 3.0*ron2)*Denom;
    h_setup->Acoef = Acoef;
    double Bcoef = 6.0*roff2*ron2*Denom;
    h_setup->Bcoef = Bcoef;
    double Ccoef = -3.0*(ron2 + roff2)*Denom;
    h_setup->Ccoef = Ccoef;
    h_setup->Constr = 2.0*Bcoef*log(roff) - Acoef/roff2 + Ccoef*roff2 + Denom*roff4;
    h_setup->Eaddr = (12.0*ron2*roff2*log(roff/ron) - 3.0*(roff4 - ron4))*Denom;
  } else {
    h_setup->Eaddr = -1.0/roff2;
  }
  
  // Constants for Gromacs-style potentials
  // Copy-pasted from Michael G. Lerner code
  h_setup->GAconst = (CT)(5.0/(3.0*roff));
  h_setup->GBcoef = (CT)(5.0/(3.0*roff2*roff2));
  double roff_ron = roff - ron;
  double roff_ron2 = roff_ron*roff_ron;
  double roff_ron3 = roff_ron*roff_ron2;
  double roff_ron4 = roff_ron2*roff_ron2;
  // NOTE: ga6, gb6, gc6 are divided by 6
  //       ga12, gb12, gc12 are divided by 12
  double ga6 = -(10.0*roff - 7.0*ron)/(roff8*roff_ron2);
  double gb6 =  ( 9.0*roff - 7.0*ron)/(roff8*roff_ron3);
  h_setup->ga6  = ga6;
  h_setup->gb6  = gb6;
  h_setup->gc6  = (1.0/roff6)/6.0 - (ga6*roff_ron3)/3.0 - (gb6*roff_ron4)/4.0;
  double ga12 = -(16.0*roff - 13.0*ron)/(roff14*roff_ron2);
  double gb12 =  (15.0*roff - 13.0*ron)/(roff14*roff_ron3);
  h_setup->ga12 = ga12;
  h_setup->gb12 = gb12;
  h_setup->gc12 = (1.0/roff12)/12.0 - (ga12*roff_ron3)/3.0 - (gb12*roff_ron4)/4.0;

  h_setup->e14fac = e14fac;

  this->vdw_model = vdw_model;
  set_elec_model(elec_model);

  set_calc_vdw(true);
  set_calc_elec(true);

  update_setup();
}

//
// Returns parameters for the nonbonded computation
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::get_setup(float& boxx, float& boxy, float& boxz, float& kappa,
					   float& roff, float& ron, float& e14fac,
					   int& vdw_model, int& elec_model) {
  boxx       = h_setup->boxx;
  boxy       = h_setup->boxy;
  boxz       = h_setup->boxz;
  kappa      = h_setup->kappa;
  roff       = h_setup->roff;
  ron        = h_setup->ron;
  e14fac     = h_setup->e14fac;
  vdw_model  = this->vdw_model;
  elec_model = this->elec_model;
}

//
// Returns box sizes
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::get_box_size(CT &boxx, CT &boxy, CT &boxz) {
  boxx = h_setup->boxx;
  boxy = h_setup->boxy;
  boxz = h_setup->boxz;
}

//
// Sets box sizes
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_box_size(const CT boxx, const CT boxy, const CT boxz) {
  h_setup->boxx = boxx;
  h_setup->boxy = boxy;
  h_setup->boxz = boxz;
  update_setup();
}

//
// Sets "calc_vdw" flag
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_calc_vdw(const bool calc_vdw) {
  this->calc_vdw = calc_vdw;
}

//
// Sets "calc_elec" flag
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_calc_elec(const bool calc_elec) {
  this->calc_elec = calc_elec;
}

//
// Returns "calc_vdw" flag
//
template <typename AT, typename CT>
bool CudaPMEDirectForce<AT, CT>::get_calc_vdw() {
  return this->calc_vdw;
}

//
// Returns "calc_elec" flag
//
template <typename AT, typename CT>
bool CudaPMEDirectForce<AT, CT>::get_calc_elec() {
  return this->calc_elec;
}

//
// Unbinds textures if they're bound
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::clearTextures() {
#ifdef USE_TEXTURE_OBJECTS
  if (vdwParamTexObjActive) {
    cudaCheck(hipDestroyTextureObject(vdwParamTexObjActive));
    vdwParamTexObjActive = false;
  }
  if (vdwParam14TexObjActive) {
    cudaCheck(hipDestroyTextureObject(vdwParam14TexObjActive));
    vdwParam14TexObjActive = false;
  }
#else
  if (get_vdwparam_texref_bound()) {
    cudaCheck(hipUnbindTexture(*get_vdwparam_texref()));
    set_vdwparam_texref_bound(false);
  }
  if (get_vdwparam14_texref_bound()) {
    cudaCheck(hipUnbindTexture(*get_vdwparam14_texref()));
    set_vdwparam14_texref_bound(false);
  }
#endif
}

//
// Sets VdW parameters by copying them from CPU
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::setup_vdwparam(const int type, const int h_nvdwparam, const CT *h_vdwparam) {
  assert(type == VDW_MAIN || type == VDW_IN14);

  int *nvdwparam_loc;
  int *vdwparam_len_loc;
  CT **vdwparam_loc;

  if (type == VDW_MAIN) {
    nvdwparam_loc = &this->nvdwparam;
    vdwparam_len_loc = &this->vdwparam_len;
    vdwparam_loc = &this->vdwparam;
  } else {
    nvdwparam_loc = &this->nvdwparam14;
    vdwparam_len_loc = &this->vdwparam14_len;
    vdwparam_loc = &this->vdwparam14;
  }

  *nvdwparam_loc = h_nvdwparam;

  // "Fix" vdwparam by multiplying c6 by 6.0 and c12 by 12.0
  // NOTE: this is done in order to avoid the multiplication in the inner loop
  CT *h_vdwparam_fixed = new CT[*nvdwparam_loc];
  for(int i=0;i < *nvdwparam_loc/2;i++) {
    h_vdwparam_fixed[i*2]   = ((CT)6.0)*h_vdwparam[i*2];
    h_vdwparam_fixed[i*2+1] = ((CT)12.0)*h_vdwparam[i*2+1];
  }

  bool vdwparam_reallocated = false;
  if (*nvdwparam_loc > *vdwparam_len_loc) {
    reallocate<CT>(vdwparam_loc, vdwparam_len_loc, *nvdwparam_loc, 1.0f);
    vdwparam_reallocated = true;
  }
  copy_HtoD_sync<CT>(h_vdwparam_fixed, *vdwparam_loc, *nvdwparam_loc);
  delete [] h_vdwparam_fixed;

  const bool *use_tex_vdwparam_loc = (type == VDW_MAIN) ? &this->use_tex_vdwparam : &this->use_tex_vdwparam14;
#ifdef USE_TEXTURE_OBJECTS
  bool *texActive = (type == VDW_MAIN) ? &vdwParamTexObjActive : &vdwParam14TexObjActive;
  hipTextureObject_t *tex = (type == VDW_MAIN) ? &vdwParamTexObj : &vdwParam14TexObj;
#else
  //bool* vdwparam_texref_bound_loc =
  //  (type == VDW_MAIN) ? get_vdwparam_texref_bound() : get_vdwparam14_texref_bound();
  texture<float2, 1, hipReadModeElementType> *vdwparam_texref_loc = 
    (type == VDW_MAIN) ? get_vdwparam_texref() : get_vdwparam14_texref();
#endif

  if (*use_tex_vdwparam_loc && vdwparam_reallocated) {
#ifdef USE_TEXTURE_OBJECTS
    if (*texActive) {
      hipDestroyTextureObject(*tex);
      *texActive = false;
    }
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = *vdwparam_loc;
    resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
    resDesc.res.linear.desc.x = sizeof(CT)*8;
    resDesc.res.linear.desc.y = sizeof(CT)*8;
    resDesc.res.linear.sizeInBytes = (*nvdwparam_loc)*sizeof(CT);

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.readMode = hipReadModeElementType;
    cudaCheck(hipCreateTextureObject(tex, &resDesc, &texDesc, NULL));
    *texActive = true;
#else
    // Unbind texture
    if ((type == VDW_MAIN) ? get_vdwparam_texref_bound() : get_vdwparam14_texref_bound()) {
      cudaCheck(hipUnbindTexture(*vdwparam_texref_loc));
      (type == VDW_MAIN) ? set_vdwparam_texref_bound(false) : set_vdwparam14_texref_bound(false);
    }
    // Bind texture
    memset(vdwparam_texref_loc, 0, sizeof(*vdwparam_texref_loc));
    vdwparam_texref_loc->normalized = 0;
    vdwparam_texref_loc->filterMode = hipFilterModePoint;
    vdwparam_texref_loc->addressMode[0] = hipAddressModeClamp;
    vdwparam_texref_loc->channelDesc.x = sizeof(CT)*8;
    vdwparam_texref_loc->channelDesc.y = sizeof(CT)*8;
    vdwparam_texref_loc->channelDesc.z = 0;
    vdwparam_texref_loc->channelDesc.w = 0;
    vdwparam_texref_loc->channelDesc.f = hipChannelFormatKindFloat;
    cudaCheck(hipBindTexture(NULL, *vdwparam_texref_loc, *vdwparam_loc, 
			      (*nvdwparam_loc)*sizeof(CT)));
    (type == VDW_MAIN) ? set_vdwparam_texref_bound(true) : set_vdwparam14_texref_bound(true);
#endif
  }
}

//
// Loads vdwparam from a file
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::load_vdwparam(const char *filename, const int nvdwparam, CT **h_vdwparam) {
  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);
    *h_vdwparam = new float[nvdwparam];
    for (int i=0;i < nvdwparam;i++) {
      file >> (*h_vdwparam)[i];
    }
    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }
}

//
// Sets VdW parameters by copying them from CPU
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwparam(const int nvdwparam, const CT *h_vdwparam) {
  setup_vdwparam(VDW_MAIN, nvdwparam, h_vdwparam);
}

//
// Sets VdW parameters by loading them from a file
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwparam(const int nvdwparam, const char *filename) {  
  CT *h_vdwparam;
  load_vdwparam(filename, nvdwparam, &h_vdwparam);
  setup_vdwparam(VDW_MAIN, nvdwparam, h_vdwparam);
  delete [] h_vdwparam;
}

//
// Sets VdW parameters by copying them from CPU
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwparam14(const int nvdwparam, const CT *h_vdwparam) {
  setup_vdwparam(VDW_IN14, nvdwparam, h_vdwparam);
}

//
// Sets VdW parameters by loading them from a file
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwparam14(const int nvdwparam, const char *filename) {  
  CT *h_vdwparam;
  load_vdwparam(filename, nvdwparam, &h_vdwparam);
  setup_vdwparam(VDW_IN14, nvdwparam, h_vdwparam);
  delete [] h_vdwparam;
}

//
// Sets vdwtype array from global list in device memory memory
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwtype(const int ncoord, const int *glo_vdwtype,
					     const int *loc2glo, hipStream_t stream) {
  // Align ncoord to warpsize
  int ncoord_aligned = ((ncoord-1)/warpsize+1)*warpsize;
  reallocate<int>(&vdwtype, &vdwtype_len, ncoord_aligned, 1.2f);

  int nthread = 512;
  int nblock = (ncoord - 1)/nthread + 1;
  set_vdwtype_kernel<<< nblock, nthread, 0, stream >>>
    (ncoord, glo_vdwtype, loc2glo, vdwtype);
  cudaCheck(hipGetLastError());
}

//
// Sets vdwtype array from host memory
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwtype(const int ncoord, const int *h_vdwtype) {
  // Align ncoord to warpsize
  int ncoord_aligned = ((ncoord-1)/warpsize+1)*warpsize;
  reallocate<int>(&vdwtype, &vdwtype_len, ncoord_aligned, 1.2f);
  copy_HtoD_sync<int>(h_vdwtype, vdwtype, ncoord);
}

//
// Sets vdwtype array by loading it from a file
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_vdwtype(const int ncoord, const char *filename) {

  int *h_vdwtype;

  std::ifstream file;
  file.exceptions(std::ifstream::failbit | std::ifstream::badbit);
  try {
    // Open file
    file.open(filename);

    h_vdwtype = new int[ncoord];

    for (int i=0;i < ncoord;i++) {
      file >> h_vdwtype[i];
    }

    file.close();
  }
  catch(std::ifstream::failure e) {
    std::cerr << "Error opening/reading/closing file " << filename << std::endl;
    exit(1);
  }

  set_vdwtype(ncoord, h_vdwtype);
  
  delete [] h_vdwtype;
}

//
// Sets 1-4 interaction and exclusion lists
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_14_list(int nin14list, int nex14list,
					     xx14list_t* h_in14list, xx14list_t* h_ex14list,
					     hipStream_t stream) {

  this->nin14list = nin14list;
  this->nex14list = nex14list;

  if (nin14list > 0) {
    reallocate<xx14list_t>(&in14list, &in14list_len, nin14list);
    copy_HtoD<xx14list_t>(h_in14list, in14list, nin14list, stream);
  }

  if (nex14list > 0) {
    reallocate<xx14list_t>(&ex14list, &ex14list_len, nex14list);
    copy_HtoD<xx14list_t>(h_ex14list, ex14list, nex14list, stream);
  }

}

//
// Setup 1-4 interaction and exclusion lists from device memory using global data:
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_14_list(const float4 *xyzq,
					     const float boxx, const float boxy, const float boxz,
					     const int *glo2loc_ind,
					     const int nin14_tbl, const int *in14_tbl, const xx14_t *in14,
					     const int nex14_tbl, const int *ex14_tbl, const xx14_t *ex14,
					     hipStream_t stream) {

  this->nin14list = nin14_tbl;
  if (nin14list > 0) reallocate<xx14list_t>(&in14list, &in14list_len, nin14list, 1.2f);

  this->nex14list = nex14_tbl;
  if (nex14list > 0) reallocate<xx14list_t>(&ex14list, &ex14list_len, nex14list, 1.2f);

  float3 half_box;
  half_box.x = boxx*0.5f;
  half_box.y = boxy*0.5f;
  half_box.z = boxz*0.5f;

  int nthread = 512;
  int nblock = (nin14_tbl + nex14_tbl - 1)/nthread + 1;
  set_14_list_kernel<<< nblock, nthread, 0, stream >>>
    (nin14_tbl, in14_tbl, in14, in14list,
     nex14_tbl, ex14_tbl, ex14, ex14list,
     xyzq, half_box, glo2loc_ind);
  cudaCheck(hipGetLastError());
}

//
// Builds Ewald lookup table
// roff  = distance cut-off
// h     = the distance between interpolation points
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::setup_ewald_force(CT h) {
  h_setup->hinv = ((CT)1.0)/h;

  n_ewald_force = (int)(sqrt(h_setup->roff2)*h_setup->hinv) + 2;

  CT *h_ewald_force = new CT[n_ewald_force];

  for (int i=1;i < n_ewald_force;i++) {
    const CT two_sqrtpi = (CT)1.12837916709551;    // 2/sqrt(pi)
    CT r = i*h;
    CT r2 = r*r;
    h_ewald_force[i] = two_sqrtpi*((CT)h_setup->kappa)*exp(-((CT)h_setup->kappa2)*r2) + 
      erfc(((CT)h_setup->kappa)*r)/r;
  }
  h_ewald_force[0] = h_ewald_force[1];

  allocate<CT>(&ewald_force, n_ewald_force);
  copy_HtoD_sync<CT>(h_ewald_force, ewald_force, n_ewald_force);

  h_setup->ewald_force = ewald_force;

  delete [] h_ewald_force;
}

//
// Sets method for calculating electrostatic force and energy
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::set_elec_model(int elec_model, CT h) {
  this->elec_model = elec_model;
  
  if (elec_model == EWALD_LOOKUP) {
    setup_ewald_force(h);
  }
}

//
// Calculates 1-4 exclusions and interactions
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::calc_14_force(const float4 *xyzq,
					       const bool calc_energy, const bool calc_virial,
					       const int stride, AT *force, hipStream_t stream) {
  if (use_tex_vdwparam14) {
#ifdef USE_TEXTURE_OBJECTS
    if (!vdwParam14TexObjActive) {
      std::cerr << "CudaPMEDirectForce<AT, CT>::calc_14_force, vdwParam14TexObj must be created" << std::endl;
      exit(1);
    }
#else
    if (!get_vdwparam14_texref_bound()) {
      std::cerr << "CudaPMEDirectForce<AT, CT>::calc_14_force, vdwparam14_texref must be bound" << std::endl;
      exit(1);
    }
#endif
  }

  int nthread = 512;
  int nin14block = (nin14list - 1)/nthread + 1;
  int nex14block = (nex14list - 1)/nthread + 1;
  int nblock = nin14block + nex14block;
  int shmem_size = 0;
  if (calc_energy) {
    shmem_size = nthread*sizeof(double2);
  }

  int vdw_model_loc = calc_vdw ? vdw_model : NONE;
  int elec_model_loc = calc_elec ? elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

  calcForce14KernelChoice<AT,CT>(nblock, nthread, shmem_size, stream,
				 vdw_model_loc, elec_model_loc, calc_energy, calc_virial,
				 this->nin14list, this->in14list, this->nex14list, this->ex14list,
				 nin14block, this->vdwtype, this->vdwparam14,
#ifdef USE_TEXTURE_OBJECTS
				 this->vdwParam14TexObj,
#endif
				 xyzq, 1.0f, stride, force,
				 energyVirial.getVirialPointer(),
				 energyVirial.getEnergyPointer(strVdw),
				 energyVirial.getEnergyPointer(strElec),
				 energyVirial.getEnergyPointer(strExcl));
}

//
// Calculates direct force
//
template <typename AT, typename CT>
void CudaPMEDirectForce<AT, CT>::calc_force(const float4 *xyzq,
					    const CudaNeighborListBuild<32>& nlist,
					    const bool calc_energy,
					    const bool calc_virial,
					    const int stride, AT *force, hipStream_t stream) {

  if (use_tex_vdwparam) {
#ifdef USE_TEXTURE_OBJECTS
    if (!vdwParamTexObjActive) {
      std::cerr << "CudaPMEDirectForce<AT, CT>::calc_force, vdwParamTexObj must be created" << std::endl;
      exit(1);
    }
#else
    if (!get_vdwparam_texref_bound()) {
      std::cerr << "CudaPMEDirectForce<AT, CT>::calc_force, vdwparam_texref must be bound" << std::endl;
      exit(1);
    }
#endif
  }

  if (nlist.get_n_ientry() == 0) return;
  int vdw_model_loc = calc_vdw ? vdw_model : NONE;
  int elec_model_loc = calc_elec ? elec_model : NONE;
  if (elec_model_loc == NONE && vdw_model_loc == NONE) return;

  int nwarp = 2;
  if (get_cuda_arch() < 300) {
    nwarp = 2;
  } else {
    nwarp = 4;
  }
  int nthread = warpsize*nwarp;
  int nblock_tot = (nlist.get_n_ientry()-1)/(nthread/warpsize)+1;

  int shmem_size = 0;
  // (sh_xi, sh_yi, sh_zi, sh_qi, sh_vdwtypei)
  if (get_cuda_arch() < 300)
    shmem_size += (nthread/warpsize)*tilesize*(sizeof(float)*4 + sizeof(int));
  // (sh_fix, sh_fiy, sh_fiz)
  shmem_size += (nthread/warpsize)*warpsize*sizeof(AT)*3;
  // If no texture fetch for vdwparam:
  if (!use_tex_vdwparam) shmem_size += nvdwparam*sizeof(float);

  if (calc_energy) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*2));
  if (calc_virial) shmem_size = max(shmem_size, (int)(nthread*sizeof(double)*3));

  calcForceKernelChoice<AT,CT>(nblock_tot, nthread, shmem_size, stream,
			       vdw_model_loc, elec_model_loc, calc_energy, calc_virial,
			       nlist, this->vdwparam, this->nvdwparam, this->vdwtype,
#ifdef USE_TEXTURE_OBJECTS
			       this->vdwParamTexObj,
#endif
			       xyzq, stride, force,
			       energyVirial.getVirialPointer(),
			       energyVirial.getEnergyPointer(strVdw),
			       energyVirial.getEnergyPointer(strElec));
}

//
// Explicit instances of CudaPMEDirectForce
//
template class CudaPMEDirectForce<long long int, float>;
