#include "hip/hip_runtime.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include "cuda_utils.h"
#include "CudaTopExcl.h"

//
// Class creator
//
CudaTopExcl::CudaTopExcl(const int ncoord, const int *iblo14, const int *inb14) : ncoord(ncoord) {
  atomExclPosLen = 0;
  atomExclPos = NULL;
  atomExclLen = 0;
  atomExcl = NULL;
  setup(iblo14, inb14);

  allocate<int>(&glo2loc, ncoord);
  set_gpu_array<int>(glo2loc, ncoord, -1);
}

//
// Class destructor
//
CudaTopExcl::~CudaTopExcl() {
  if (atomExclPos != NULL) deallocate<int>(&atomExclPos);
  if (atomExcl != NULL) deallocate<int>(&atomExcl);
  deallocate<int>(&glo2loc);
}

//
// Setups topological exclusions from data structure used in CHARMM
//
void CudaTopExcl::setup(const int *iblo14, const int *inb14) {

  int *nexcl = new int[ncoord];

  for (int i=0;i < ncoord;i++) nexcl[i] = 0;

  // Count the number of exclusions to nexcl[0 ... ncoord-1]
  for (int i=0;i < ncoord;i++) {
    int excl_start;
    if (i > 0) {
      excl_start = iblo14[i-1];
    } else {
      excl_start = 0;
    }
    int excl_end = iblo14[i] - 1;
    // add i-j exclusions to atom i
    nexcl[i] += excl_end - excl_start + 1;
    for (int excl_i=excl_start; excl_i <= excl_end;excl_i++) {
      int j = abs(inb14[excl_i]) - 1;
      // add i-j exclusion to atom j
      nexcl[j]++;
    }
  }

  // Find out maximum number of atom-atom exclusions per atom
  maxNumExcl = 0;
  for (int i=0;i < ncoord;i++) maxNumExcl = max(maxNumExcl, nexcl[i]);  

  int *h_atomExclPos = new int[ncoord+1];

  // Use exclusive cumulative sum to calculate positions
  int nexcl_tot = 0;
  for (int i=0;i < ncoord;i++) {
    h_atomExclPos[i] = nexcl_tot;
    nexcl_tot += nexcl[i];
  }
  h_atomExclPos[ncoord] = nexcl_tot;

  int *h_atomExcl = new int[nexcl_tot];

  for (int i=0;i < ncoord;i++) nexcl[i] = 0;

  for (int i=0;i < ncoord;i++) {
    int excl_start;
    if (i > 0) {
      excl_start = iblo14[i-1];
    } else {
      excl_start = 0;
    }
    int excl_end = iblo14[i] - 1;

    int pos_starti = h_atomExclPos[i];
    int ni = nexcl[i];
    for (int excl_i=excl_start;excl_i <= excl_end;excl_i++) {
      int j = abs(inb14[excl_i]) - 1;
      // Add i-j exclusion to atom j
      int pos_startj = h_atomExclPos[j];
      int nj = nexcl[j];
      if (pos_startj + nj >= h_atomExclPos[j+1]) {
	std::cerr << "CudaTopExcl::setup, overflow in j" << std::endl;
	exit(1);
      }
      h_atomExcl[pos_startj + nj] = i;
      nj++;
      nexcl[j] = nj;
      // Add i-j exclusion to atom i
      if (pos_starti + ni >= h_atomExclPos[i+1]) {
	std::cerr << "CudaTopExcl::setup, overflow in i" << std::endl;
	exit(1);
      }
      h_atomExcl[pos_starti + ni] = j;
      ni++;
    }

    nexcl[i] = ni;
  }

  // Allocate GPU memory and copy results to GPU
#ifdef STRICT_MEMORY_REALLOC
  reallocate<int>(&atomExclPos, &atomExclPosLen, ncoord+1, 1.0f);
  reallocate<int>(&atomExcl, &atomExclLen, nexcl_tot, 1.0f);
#else
  reallocate<int>(&atomExclPos, &atomExclPosLen, ncoord+1, 1.1f);
  reallocate<int>(&atomExcl, &atomExclLen, nexcl_tot, 1.1f);
#endif
  copy_HtoD_sync<int>(h_atomExclPos, atomExclPos, ncoord+1);
  copy_HtoD_sync<int>(h_atomExcl, atomExcl, nexcl_tot);
  
  delete [] h_atomExcl;
  delete [] h_atomExclPos;
  delete [] nexcl;
}
