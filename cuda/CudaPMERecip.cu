#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <math.h>
#include "gpu_utils.h"
#include "cuda_utils.h"
#include "reduce.h"
#include "CudaPMERecip.h"

static const double pi = 3.14159265358979323846;

//
// CudaPMERecip class
//
// AT  = Accumulation Type
// CT  = Calculation Type (real)
// CT2 = Calculation Type (complex)
//
// (c) Antti-Pekka Hynninen, 2013, aphynninen@hotmail.com
//
// In real space:
// Each instance of CudaPMERecip is responsible for grid region (x0..x1) x (y0..y1) x (z0..z1)
// Note that usually x0=0, x1=nfftx-1
//

template <typename T>
__forceinline__ __device__ void write_grid(const float val, const int ind,
					   T* data) {
  // The generic version can not be used
}

// Template specialization for 64bit integer = "long long int"
template <>
__forceinline__ __device__ void write_grid <long long int> (const float val,
							    const int ind,
							    long long int* data) {
  unsigned long long int qintp = llitoulli(lliroundf(FORCE_SCALE*val));
  atomicAdd((unsigned long long int *)&data[ind], qintp);
}

// Template specialization for 32bit integer = "int"
template <>
__forceinline__ __device__ void write_grid <int> (const float val,
						  const int ind,
						  int* data) {
  unsigned int qintp = itoui(iroundf(FORCE_SCALE_I*val));
  atomicAdd((unsigned int *)&data[ind], qintp);
}

/*
//
// Temporary kernels that change the data layout
//
__global__ void change_gridp(const int ncoord, const gridp_t *gridp,
			     int *ixtbl, int *iytbl, int *iztbl, float *charge) {

  unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  if (pos < ncoord) {
    gridp_t gridpval = gridp[pos];
    int x = gridpval.x;
    int y = gridpval.y;
    int z = gridpval.z;
    float q = gridpval.q;
    
    ixtbl[pos] = x;
    iytbl[pos] = y;
    iztbl[pos] = z;
    charge[pos] = q;
  }

}
*/

/*
__global__ void change_theta(const int ncoord, const float3 *theta,
			     float4 *thetax, float4 *thetay, float4 *thetaz) {

  unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  if (pos < ncoord) {
  thetax[pos].x = theta[pos*4].x;
    thetax[pos].y = theta[pos*4+1].x;
    thetax[pos].z = theta[pos*4+2].x;
    thetax[pos].w = theta[pos*4+3].x;

    thetay[pos].x = theta[pos*4].y;
    thetay[pos].y = theta[pos*4+1].y;
    thetay[pos].z = theta[pos*4+2].y;
    thetay[pos].w = theta[pos*4+3].y;

    thetaz[pos].x = theta[pos*4].z;
    thetaz[pos].y = theta[pos*4+1].z;
    thetaz[pos].z = theta[pos*4+2].z;
    thetaz[pos].w = theta[pos*4+3].z;    
  }

}
*/

//
// Data structure for spread_charge -kernels
//
struct spread_t {
  int ix;
  int iy;
  int iz;
  float thetax[4];
  float thetay[4];
  float thetaz[4];
};

//
// Spreads the charge on the grid
// blockDim.x               = Number of atoms each block loads
// blockDim.y*blockDim.x/64 = Number of atoms we spread at once
//
template <typename AT>
__global__ void
spread_charge_4(const int ncoord,
		const int *ixtbl, const int *iytbl, const int *iztbl, const float *charge,
		const float4 *thetax, const float4 *thetay, const float4 *thetaz,
		const int nfftx, const int nffty, const int nfftz,
		AT* data) {

  // Shared memory
  extern __shared__ spread_t shmem[];

  // Process atoms pos to pos_end-1
  unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end) {
    int ix = ixtbl[pos];
    int iy = iytbl[pos];
    int iz = iztbl[pos];
    float q = charge[pos];
    // For each atom we write 4x4x4=64 grid points
    // 3*4 = 12 values per atom stored:
    // theta_sh[i].x = theta_x for grid point i=0...3
    // theta_sh[i].y = theta_y for grid point i=0...3
    // theta_sh[i].z = theta_z for grid point i=0...3
    float4 thetax_tmp = thetax[pos];
    float4 thetay_tmp = thetay[pos];
    float4 thetaz_tmp = thetaz[pos];

    thetax_tmp.x *= q;
    thetax_tmp.y *= q;
    thetax_tmp.z *= q;
    thetax_tmp.w *= q;

    shmem[threadIdx.x].ix = ix;
    shmem[threadIdx.x].iy = iy;
    shmem[threadIdx.x].iz = iz;
    
    shmem[threadIdx.x].thetax[0] = thetax_tmp.x;
    shmem[threadIdx.x].thetax[1] = thetax_tmp.y;
    shmem[threadIdx.x].thetax[2] = thetax_tmp.z;
    shmem[threadIdx.x].thetax[3] = thetax_tmp.w;
    
    shmem[threadIdx.x].thetay[0] = thetay_tmp.x;
    shmem[threadIdx.x].thetay[1] = thetay_tmp.y;
    shmem[threadIdx.x].thetay[2] = thetay_tmp.z;
    shmem[threadIdx.x].thetay[3] = thetay_tmp.w;

    shmem[threadIdx.x].thetaz[0] = thetaz_tmp.x;
    shmem[threadIdx.x].thetaz[1] = thetaz_tmp.y;
    shmem[threadIdx.x].thetaz[2] = thetaz_tmp.z;
    shmem[threadIdx.x].thetaz[3] = thetaz_tmp.w;

  }
  __syncthreads();

  // Grid point location, values of (ix0, iy0, iz0) are in range 0..3
  const int tid = (threadIdx.x + threadIdx.y*blockDim.x) % 64;
  const int x0 = tid & 3;
  const int y0 = (tid >> 2) & 3;
  const int z0 = tid >> 4;

  // Loop over atoms pos..pos_end-1
  int iadd = blockDim.x*blockDim.y/64;
  int i = (threadIdx.x + threadIdx.y*blockDim.x)/64;
  int iend = pos_end - blockIdx.x*blockDim.x;
  for (;i < iend;i += iadd) {
    int x = shmem[i].ix + x0;
    int y = shmem[i].iy + y0;
    int z = shmem[i].iz + z0;
      
    if (x >= nfftx) x -= nfftx;
    if (y >= nffty) y -= nffty;
    if (z >= nfftz) z -= nfftz;
      
    // Get position on the grid
    int ind = x + nfftx*(y + nffty*z);
      
    // Here we unroll the 4x4x4 loop with 64 threads
    // Calculate interpolated charge value and store it to global memory
    write_grid<AT>(shmem[i].thetax[x0]*shmem[i].thetay[y0]*shmem[i].thetaz[z0], ind, data);

  }

}

//
// Calculate theta and dtheta for order 4 bspline
//
template <typename T, typename T3>
__forceinline__ __device__ void calc_theta_dtheta_4(T wx, T wy, T wz, T3 *theta_tmp, T3 *dtheta_tmp) {
  theta_tmp[3].x = ((T)0);
  theta_tmp[3].y = ((T)0);
  theta_tmp[3].z = ((T)0);
  theta_tmp[1].x = wx;
  theta_tmp[1].y = wy;
  theta_tmp[1].z = wz;
  theta_tmp[0].x = ((T)1) - wx;
  theta_tmp[0].y = ((T)1) - wy;
  theta_tmp[0].z = ((T)1) - wz;
  
  // compute standard b-spline recursion
  theta_tmp[2].x = ((T)0.5)*wx*theta_tmp[1].x;
  theta_tmp[2].y = ((T)0.5)*wy*theta_tmp[1].y;
  theta_tmp[2].z = ((T)0.5)*wz*theta_tmp[1].z;
       
  theta_tmp[1].x = ((T)0.5)*((wx+((T)1.0))*theta_tmp[0].x + (((T)2.0)-wx)*theta_tmp[1].x);
  theta_tmp[1].y = ((T)0.5)*((wy+((T)1.0))*theta_tmp[0].y + (((T)2.0)-wy)*theta_tmp[1].y);
  theta_tmp[1].z = ((T)0.5)*((wz+((T)1.0))*theta_tmp[0].z + (((T)2.0)-wz)*theta_tmp[1].z);
       
  theta_tmp[0].x = ((T)0.5)*(((T)1.0)-wx)*theta_tmp[0].x;
  theta_tmp[0].y = ((T)0.5)*(((T)1.0)-wy)*theta_tmp[0].y;
  theta_tmp[0].z = ((T)0.5)*(((T)1.0)-wz)*theta_tmp[0].z;
       
  // perform standard b-spline differentiationa
  dtheta_tmp[0].x = -theta_tmp[0].x;
  dtheta_tmp[0].y = -theta_tmp[0].y;
  dtheta_tmp[0].z = -theta_tmp[0].z;

  dtheta_tmp[1].x = theta_tmp[0].x - theta_tmp[1].x;
  dtheta_tmp[1].y = theta_tmp[0].y - theta_tmp[1].y;
  dtheta_tmp[1].z = theta_tmp[0].z - theta_tmp[1].z;

  dtheta_tmp[2].x = theta_tmp[1].x - theta_tmp[2].x;
  dtheta_tmp[2].y = theta_tmp[1].y - theta_tmp[2].y;
  dtheta_tmp[2].z = theta_tmp[1].z - theta_tmp[2].z;

  dtheta_tmp[3].x = theta_tmp[2].x - theta_tmp[3].x;
  dtheta_tmp[3].y = theta_tmp[2].y - theta_tmp[3].y;
  dtheta_tmp[3].z = theta_tmp[2].z - theta_tmp[3].z;
          
  // one more recursion
  theta_tmp[3].x = (((T)1.0)/((T)3.0))*wx*theta_tmp[2].x;
  theta_tmp[3].y = (((T)1.0)/((T)3.0))*wy*theta_tmp[2].y;
  theta_tmp[3].z = (((T)1.0)/((T)3.0))*wz*theta_tmp[2].z;
  
  theta_tmp[2].x = (((T)1.0)/((T)3.0))*((wx+((T)1.0))*theta_tmp[1].x + (((T)3.0)-wx)*theta_tmp[2].x);
  theta_tmp[2].y = (((T)1.0)/((T)3.0))*((wy+((T)1.0))*theta_tmp[1].y + (((T)3.0)-wy)*theta_tmp[2].y);
  theta_tmp[2].z = (((T)1.0)/((T)3.0))*((wz+((T)1.0))*theta_tmp[1].z + (((T)3.0)-wz)*theta_tmp[2].z);
  
  theta_tmp[1].x = (((T)1.0)/((T)3.0))*((wx+((T)2.0))*theta_tmp[0].x + (((T)2.0)-wx)*theta_tmp[1].x);
  theta_tmp[1].y = (((T)1.0)/((T)3.0))*((wy+((T)2.0))*theta_tmp[0].y + (((T)2.0)-wy)*theta_tmp[1].y);
  theta_tmp[1].z = (((T)1.0)/((T)3.0))*((wz+((T)2.0))*theta_tmp[0].z + (((T)2.0)-wz)*theta_tmp[1].z);
  
  theta_tmp[0].x = (((T)1.0)/((T)3.0))*(((T)1.0)-wx)*theta_tmp[0].x;
  theta_tmp[0].y = (((T)1.0)/((T)3.0))*(((T)1.0)-wy)*theta_tmp[0].y;
  theta_tmp[0].z = (((T)1.0)/((T)3.0))*(((T)1.0)-wz)*theta_tmp[0].z;
}

//
// Calculate theta and dtheta for general order bspline
//
template <typename T, typename T3, int order>
__forceinline__ __device__ void calc_theta_dtheta(T wx, T wy, T wz, T3 *theta, T3 *dtheta) {

  theta[order-1].x = ((T)0);
  theta[order-1].y = ((T)0);
  theta[order-1].z = ((T)0);
  theta[1].x = wx;
  theta[1].y = wy;
  theta[1].z = wz;
  theta[0].x = ((T)1) - wx;
  theta[0].y = ((T)1) - wy;
  theta[0].z = ((T)1) - wz;

#pragma unroll
  for (int k=3;k <= order-1;k++) {
    T div = ((T)1) / (T)(k-1);
    theta[k-1].x = div*wx*theta[k-2].x;
    theta[k-1].y = div*wy*theta[k-2].y;
    theta[k-1].z = div*wz*theta[k-2].z;
#pragma unroll
    for (int j=1;j <= k-2;j++) {
      theta[k-j-1].x = div*((wx + j)*theta[k-j-2].x + (k-j-wx)*theta[k-j-1].x);
      theta[k-j-1].y = div*((wy + j)*theta[k-j-2].y + (k-j-wy)*theta[k-j-1].y);
      theta[k-j-1].z = div*((wz + j)*theta[k-j-2].z + (k-j-wz)*theta[k-j-1].z);
    }
    theta[0].x = div*(((T)1) - wx)*theta[0].x;
    theta[0].y = div*(((T)1) - wy)*theta[0].y;
    theta[0].z = div*(((T)1) - wz)*theta[0].z;
  }

  //--- perform standard b-spline differentiation
  dtheta[0].x = -theta[0].x;
  dtheta[0].y = -theta[0].y;
  dtheta[0].z = -theta[0].z;
#pragma unroll
  for (int j=2;j <= order;j++) {
    dtheta[j-1].x = theta[j-2].x - theta[j-1].x;
    dtheta[j-1].y = theta[j-2].y - theta[j-1].y;
    dtheta[j-1].z = theta[j-2].z - theta[j-1].z;
  }
	    
  //--- one more recursion
  T div = ((T)1) / (T)(order-1);
  theta[order-1].x = div*wx*theta[order-2].x;
  theta[order-1].y = div*wy*theta[order-2].y;
  theta[order-1].z = div*wz*theta[order-2].z;
#pragma unroll
  for (int j=1;j <= order-2;j++) {
    theta[order-j-1].x = div*((wx + j)*theta[order-j-2].x + (order-j-wx)*theta[order-j-1].x);
    theta[order-j-1].y = div*((wy + j)*theta[order-j-2].y + (order-j-wy)*theta[order-j-1].y);
    theta[order-j-1].z = div*((wz + j)*theta[order-j-2].z + (order-j-wz)*theta[order-j-1].z);
  }
    
  theta[0].x = div*(((T)1) - wx)*theta[0].x;
  theta[0].y = div*(((T)1) - wy)*theta[0].y;
  theta[0].z = div*(((T)1) - wz)*theta[0].z;
}

//
// Calculate theta and dtheta for order 4 bspline
//
template <typename T>
__forceinline__ __device__ void calc_one_theta_dtheta_4(T w, T theta_tmp[4], T dtheta_tmp[4]) {
  theta_tmp[3] = ((T)0);
  theta_tmp[1] = w;
  theta_tmp[0] = ((T)1) - w;
  
  // compute standard b-spline recursion
  theta_tmp[2] = ((T)0.5)*w*theta_tmp[1];
  theta_tmp[1] = ((T)0.5)*((w+((T)1.0))*theta_tmp[0] + (((T)2.0)-w)*theta_tmp[1]);
  theta_tmp[0] = ((T)0.5)*(((T)1.0)-w)*theta_tmp[0];
       
  // perform standard b-spline differentiationa
  dtheta_tmp[0] = -theta_tmp[0];
  dtheta_tmp[1] = theta_tmp[0] - theta_tmp[1];
  dtheta_tmp[2] = theta_tmp[1] - theta_tmp[2];
  dtheta_tmp[3] = theta_tmp[2] - theta_tmp[3];
          
  // one more recursion
  theta_tmp[3] = (((T)1.0)/((T)3.0))*w*theta_tmp[2];
  theta_tmp[2] = (((T)1.0)/((T)3.0))*((w+((T)1.0))*theta_tmp[1] + (((T)3.0)-w)*theta_tmp[2]);
  theta_tmp[1] = (((T)1.0)/((T)3.0))*((w+((T)2.0))*theta_tmp[0] + (((T)2.0)-w)*theta_tmp[1]);
  theta_tmp[0] = (((T)1.0)/((T)3.0))*(((T)1.0)-w)*theta_tmp[0];
}

template <typename T>
__forceinline__ __device__ void calc_one_theta_dtheta_4b(T w, T &theta0, T &theta1, T &theta2, T &theta3,
							T &dtheta0, T &dtheta1, T &dtheta2, T &dtheta3) {

  theta3 = ((T)0);
  theta1 = w;
  theta0 = ((T)1) - w;
  
  // compute standard b-spline recursion
  theta2 = ((T)0.5)*w*theta1;
  theta1 = ((T)0.5)*((w+((T)1.0))*theta0 + (((T)2.0)-w)*theta1);
  theta0 = ((T)0.5)*(((T)1.0)-w)*theta0;
  
  // perform standard b-spline differentiationa
  dtheta0 = -theta0;
  dtheta1 = theta0 - theta1;
  dtheta2 = theta1 - theta2;
  dtheta3 = theta2 - theta3;
      
  // one more recursion
  theta3 = (((T)1.0)/((T)3.0))*w*theta2;
  theta2 = (((T)1.0)/((T)3.0))*((w+((T)1.0))*theta1 + (((T)3.0)-w)*theta2);
  theta1 = (((T)1.0)/((T)3.0))*((w+((T)2.0))*theta0 + (((T)2.0)-w)*theta1);
  theta0 = (((T)1.0)/((T)3.0))*(((T)1.0)-w)*theta0;
}

template <typename T>
__forceinline__ __device__ void calc_one_theta_4(T w, T &theta0, T &theta1, T &theta2, T &theta3) {

  theta3 = ((T)0);
  theta1 = w;
  theta0 = ((T)1) - w;
  
  // compute standard b-spline recursion
  theta2 = ((T)0.5)*w*theta1;
  theta1 = ((T)0.5)*((w+((T)1.0))*theta0 + (((T)2.0)-w)*theta1);
  theta0 = ((T)0.5)*(((T)1.0)-w)*theta0;
  
  // one more recursion
  theta3 = (((T)1.0)/((T)3.0))*w*theta2;
  theta2 = (((T)1.0)/((T)3.0))*((w+((T)1.0))*theta1 + (((T)3.0)-w)*theta2);
  theta1 = (((T)1.0)/((T)3.0))*((w+((T)2.0))*theta0 + (((T)2.0)-w)*theta1);
  theta0 = (((T)1.0)/((T)3.0))*(((T)1.0)-w)*theta0;
}

//
// General version for any order
//
template <typename T, int order>
__forceinline__ __device__ void calc_one_theta(const T w, T *theta) {

  theta[order-1] = ((T)0);
  theta[1] = w;
  theta[0] = ((T)1) - w;

#pragma unroll
  for (int k=3;k <= order-1;k++) {
    T div = ((T)1) / (T)(k-1);
    theta[k-1] = div*w*theta[k-2];
#pragma unroll
    for (int j=1;j <= k-2;j++) {
      theta[k-j-1] = div*((w+j)*theta[k-j-2] + (k-j-w)*theta[k-j-1]);
    }
    theta[0] = div*(((T)1) - w)*theta[0];
  }
	    
  //--- one more recursion
  T div = ((T)1) / (T)(order-1);
  theta[order-1] = div*w*theta[order-2];
#pragma unroll
  for (int j=1;j <= order-2;j++) {
    theta[order-j-1] = div*((w+j)*theta[order-j-2] + (order-j-w)*theta[order-j-1]);
  }
    
  theta[0] = div*(((T)1) - w)*theta[0];
}

//
// Spreads the charge on the grid. Calculates theta and dtheta on the fly
// blockDim.x               = Number of atoms each block loads
// blockDim.y*blockDim.x/64 = Number of atoms we spread at once
//
template <typename AT>
__global__ void
spread_charge_ortho_4(const float4 *xyzq, const int ncoord,
		      const float recip11, const float recip22, const float recip33,
		      const int nfftx, const int nffty, const int nfftz,
		      float *thetax, float *thetay, float *thetaz,
		      float *dthetax, float *dthetay, float *dthetaz,
		      AT* data) {
  
  // Shared memory
  //extern __shared__ void shmem[];

  __shared__ int sh_ix[32];
  __shared__ int sh_iy[32];
  __shared__ int sh_iz[32];
  __shared__ float sh_q[32];
  __shared__ float sh_thetax[4*32];
  __shared__ float sh_thetay[4*32];
  __shared__ float sh_thetaz[4*32];
  __shared__ float sh_dthetax[4*32];
  __shared__ float sh_dthetay[4*32];
  __shared__ float sh_dthetaz[4*32];

  // Process atoms pos to pos_end-1
  const unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end) {

    float recip;
    int nfft;
    float x;

    float4 xyzqi = xyzq[pos];

    if (threadIdx.y == 0) {
      recip = recip11;
      nfft = nfftx;
      x = xyzqi.x;
    } else if (threadIdx.y == 1) {
      recip = recip22;
      nfft = nffty;
      x = xyzqi.y;
    } else if (threadIdx.y == 2) {
      recip = recip33;
      nfft = nfftz;
      x = xyzqi.z;
    } else {
      x = xyzqi.w;
    }

    int fri;
    float theta0, theta1, theta2, theta3;
    float dtheta0, dtheta1, dtheta2, dtheta3;

    if (threadIdx.y < 3) {
      float w, fr;
      w = x*recip + 2.0f;
      fr = (float)(nfft*(w - (floorf(w + 0.5f) - 0.5f)));
      fri = (int)fr;
      w = fr - (float)fri;

      calc_one_theta_dtheta_4b<float>(w, theta0, theta1, theta2, theta3,
				      dtheta0, dtheta1, dtheta2, dtheta3);
    }
    
    if (threadIdx.y == 0) {
      sh_ix[threadIdx.x] = fri;
      sh_thetax[threadIdx.x*4 + 0] = theta0;
      sh_thetax[threadIdx.x*4 + 1] = theta1;
      sh_thetax[threadIdx.x*4 + 2] = theta2;
      sh_thetax[threadIdx.x*4 + 3] = theta3;
      sh_dthetax[threadIdx.x*4 + 0] = dtheta0;
      sh_dthetax[threadIdx.x*4 + 1] = dtheta1;
      sh_dthetax[threadIdx.x*4 + 2] = dtheta2;
      sh_dthetax[threadIdx.x*4 + 3] = dtheta3;
    } else if (threadIdx.y == 1) {
      sh_iy[threadIdx.x] = fri;
      sh_thetay[threadIdx.x*4 + 0] = theta0;
      sh_thetay[threadIdx.x*4 + 1] = theta1;
      sh_thetay[threadIdx.x*4 + 2] = theta2;
      sh_thetay[threadIdx.x*4 + 3] = theta3;
      sh_dthetay[threadIdx.x*4 + 0] = dtheta0;
      sh_dthetay[threadIdx.x*4 + 1] = dtheta1;
      sh_dthetay[threadIdx.x*4 + 2] = dtheta2;
      sh_dthetay[threadIdx.x*4 + 3] = dtheta3;
    } else if (threadIdx.y == 2) {
      sh_iz[threadIdx.x] = fri;
      sh_thetaz[threadIdx.x*4 + 0] = theta0;
      sh_thetaz[threadIdx.x*4 + 1] = theta1;
      sh_thetaz[threadIdx.x*4 + 2] = theta2;
      sh_thetaz[threadIdx.x*4 + 3] = theta3;
      sh_dthetaz[threadIdx.x*4 + 0] = dtheta0;
      sh_dthetaz[threadIdx.x*4 + 1] = dtheta1;
      sh_dthetaz[threadIdx.x*4 + 2] = dtheta2;
      sh_dthetaz[threadIdx.x*4 + 3] = dtheta3;
    } else {
      sh_q[threadIdx.x] = x;
    }


  }

  __syncthreads();

  // Write to global memory
  if (pos < pos_end) {
    const int t = threadIdx.x + blockDim.x*threadIdx.y; // 0...127
    const int pos0 = blockIdx.x*blockDim.x*blockDim.y;  // 0, 128, 256, ...
    thetax[pos0 + t] = sh_thetax[t];
    thetay[pos0 + t] = sh_thetay[t];
    thetaz[pos0 + t] = sh_thetaz[t];
    dthetax[pos0 + t] = sh_dthetax[t];
    dthetay[pos0 + t] = sh_dthetay[t];
    dthetaz[pos0 + t] = sh_dthetaz[t];    
  }

  // Grid point location, values of (ix0, iy0, iz0) are in range 0..3
  const int tid = (threadIdx.x + threadIdx.y*blockDim.x) % 64;
  const int x0 = tid & 3;
  const int y0 = (tid >> 2) & 3;
  const int z0 = tid >> 4;

  // Loop over atoms pos..pos_end-1
  int iadd = blockDim.x*blockDim.y/64;
  int i = (threadIdx.x + threadIdx.y*blockDim.x)/64;
  int iend = pos_end - blockIdx.x*blockDim.x;
  for (;i < iend;i += iadd) {
    int x = sh_ix[i] + x0;
    int y = sh_iy[i] + y0;
    int z = sh_iz[i] + z0;
    float q = sh_q[i];
      
    if (x >= nfftx) x -= nfftx;
    if (y >= nffty) y -= nffty;
    if (z >= nfftz) z -= nfftz;
      
    // Get position on the grid
    int ind = x + nfftx*(y + nffty*z);
      
    // Here we unroll the 4x4x4 loop with 64 threads
    // Calculate interpolated charge value and store it to global memory
    write_grid<AT>(q*sh_thetax[i*4+x0]*sh_thetay[i*4+y0]*sh_thetaz[i*4+z0], ind, data);

  }

}

//
// Spreads the charge on the grid. Calculates theta and dtheta on the fly
// blockDim.x               = Number of atoms each block loads
// blockDim.y*blockDim.x/64 = Number of atoms we spread at once
//
template <typename AT>
__global__ void
spread_charge_ortho_4(const float4 *xyzq, const int ncoord,
		      const float recip11, const float recip22, const float recip33,
		      const int nfftx, const int nffty, const int nfftz,
		      AT* data) {
  
  // Shared memory. Uses 2048 bytes
  __shared__ int sh_ix[32];
  __shared__ int sh_iy[32];
  __shared__ int sh_iz[32];
  __shared__ float sh_q[32];
  __shared__ float sh_thetax[4*32];
  __shared__ float sh_thetay[4*32];
  __shared__ float sh_thetaz[4*32];

  // Process atoms pos to pos_end-1
  const unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    sh_q[threadIdx.x] = q;

    float w;

    w = x*recip11 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));
    w = y*recip22 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));
    w = z*recip33 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    sh_ix[threadIdx.x] = frxi;
    sh_iy[threadIdx.x] = fryi;
    sh_iz[threadIdx.x] = frzi;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float theta0, theta1, theta2, theta3;

    calc_one_theta_4<float>(wx, theta0, theta1, theta2, theta3);
    sh_thetax[threadIdx.x*4 + 0] = theta0;
    sh_thetax[threadIdx.x*4 + 1] = theta1;
    sh_thetax[threadIdx.x*4 + 2] = theta2;
    sh_thetax[threadIdx.x*4 + 3] = theta3;

    calc_one_theta_4<float>(wy, theta0, theta1, theta2, theta3);
    sh_thetay[threadIdx.x*4 + 0] = theta0;
    sh_thetay[threadIdx.x*4 + 1] = theta1;
    sh_thetay[threadIdx.x*4 + 2] = theta2;
    sh_thetay[threadIdx.x*4 + 3] = theta3;

    calc_one_theta_4<float>(wz, theta0, theta1, theta2, theta3);
    sh_thetaz[threadIdx.x*4 + 0] = theta0;
    sh_thetaz[threadIdx.x*4 + 1] = theta1;
    sh_thetaz[threadIdx.x*4 + 2] = theta2;
    sh_thetaz[threadIdx.x*4 + 3] = theta3;

  }

  __syncthreads();

  // Grid point location, values of (ix0, iy0, iz0) are in range 0..3
  const int tid = (threadIdx.x + threadIdx.y*blockDim.x) % 64;   // 0...63
  const int x0 = tid & 3;
  const int y0 = (tid >> 2) & 3;
  const int z0 = tid >> 4;

  // Loop over atoms pos..pos_end-1
  int iadd = blockDim.x*blockDim.y/64;
  int i = (threadIdx.x + threadIdx.y*blockDim.x)/64;
  int iend = pos_end - blockIdx.x*blockDim.x;
  for (;i < iend;i += iadd) {
    int x = sh_ix[i] + x0;
    int y = sh_iy[i] + y0;
    int z = sh_iz[i] + z0;
    float q = sh_q[i];
      
    if (x >= nfftx) x -= nfftx;
    if (y >= nffty) y -= nffty;
    if (z >= nfftz) z -= nfftz;
      
    // Get position on the grid
    int ind = x + nfftx*(y + nffty*z);
      
    // Here we unroll the 4x4x4 loop with 64 threads
    // Calculate interpolated charge value and store it to global memory
    write_grid<AT>(q*sh_thetax[i*4+x0]*sh_thetay[i*4+y0]*sh_thetaz[i*4+z0], ind, data);

  }

}

//
// Spreads the charge on the grid. Calculates theta and dtheta on the fly
// blockDim.x               = Number of atoms each block loads
// blockDim.y*blockDim.x/64 = Number of atoms we spread at once
//
template <typename AT>
__global__ void
spread_charge_ortho_6(const float4 *xyzq, const int ncoord,
		      const float recip11, const float recip22, const float recip33,
		      const int nfftx, const int nffty, const int nfftz,
		      AT* data) {
  // Uses 2816 bytes of shared memory
  __shared__ int sh_ix[32];
  __shared__ int sh_iy[32];
  __shared__ int sh_iz[32];
  __shared__ float sh_q[32];
  __shared__ float sh_thetax[6*32];
  __shared__ float sh_thetay[6*32];
  __shared__ float sh_thetaz[6*32];

  // Process atoms pos to pos_end-1
  const unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    sh_q[threadIdx.x] = q;

    float w;

    w = x*recip11 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));
    w = y*recip22 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));
    w = z*recip33 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    sh_ix[threadIdx.x] = frxi;
    sh_iy[threadIdx.x] = fryi;
    sh_iz[threadIdx.x] = frzi;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float theta[6];

    calc_one_theta<float, 6>(wx, theta);
    sh_thetax[threadIdx.x*6 + 0] = theta[0];
    sh_thetax[threadIdx.x*6 + 1] = theta[1];
    sh_thetax[threadIdx.x*6 + 2] = theta[2];
    sh_thetax[threadIdx.x*6 + 3] = theta[3];
    sh_thetax[threadIdx.x*6 + 4] = theta[4];
    sh_thetax[threadIdx.x*6 + 5] = theta[5];

    calc_one_theta<float, 6>(wy, theta);
    sh_thetay[threadIdx.x*6 + 0] = theta[0];
    sh_thetay[threadIdx.x*6 + 1] = theta[1];
    sh_thetay[threadIdx.x*6 + 2] = theta[2];
    sh_thetay[threadIdx.x*6 + 3] = theta[3];
    sh_thetay[threadIdx.x*6 + 4] = theta[4];
    sh_thetay[threadIdx.x*6 + 5] = theta[5];

    calc_one_theta<float, 6>(wz, theta);
    sh_thetaz[threadIdx.x*6 + 0] = theta[0];
    sh_thetaz[threadIdx.x*6 + 1] = theta[1];
    sh_thetaz[threadIdx.x*6 + 2] = theta[2];
    sh_thetaz[threadIdx.x*6 + 3] = theta[3];
    sh_thetaz[threadIdx.x*6 + 4] = theta[4];
    sh_thetaz[threadIdx.x*6 + 5] = theta[5];

  }

  __syncthreads();

  // Grid point location, values of (ix0, iy0, iz0) are in range 0..5
  // NOTE: Only tid=0...215 do any computation
  const int tid = (threadIdx.x + threadIdx.y*blockDim.x) % 224;   // 0...223
  const int x0 = tid % 6;
  const int y0 = (tid / 6) % 6;
  const int z0 = tid / 36;

  // Loop over atoms pos..pos_end-1
  int iadd = blockDim.x*blockDim.y/224;
  int i = (threadIdx.x + threadIdx.y*blockDim.x)/224;
  int iend = pos_end - blockIdx.x*blockDim.x;
  for (;i < iend;i += iadd) {
    int x = sh_ix[i] + x0;
    int y = sh_iy[i] + y0;
    int z = sh_iz[i] + z0;
    float q = sh_q[i];
      
    if (x >= nfftx) x -= nfftx;
    if (y >= nffty) y -= nffty;
    if (z >= nfftz) z -= nfftz;
      
    // Get position on the grid
    int ind = x + nfftx*(y + nffty*z);
      
    // Here we unroll the 6x6x6 loop with 216 threads.
    // NOTE: We use 7*32=224 threads to do this
    // Calculate interpolated charge value and store it to global memory
    if (tid < 216) write_grid<AT>(q*sh_thetax[i*6+x0]*sh_thetay[i*6+y0]*sh_thetaz[i*6+z0], ind, data);
  }

}

//
// Spreads the charge on the grid. Calculates theta and dtheta on the fly
// blockDim.x               = Number of atoms each block loads
// blockDim.y*blockDim.x/64 = Number of atoms we spread at once
//
template <typename AT>
__global__ void
spread_charge_ortho_8(const float4 *xyzq, const int ncoord,
		      const float recip11, const float recip22, const float recip33,
		      const int nfftx, const int nffty, const int nfftz,
		      AT* data) {

  // Uses 3584 bytes of shared memory
  __shared__ int sh_ix[32];
  __shared__ int sh_iy[32];
  __shared__ int sh_iz[32];
  __shared__ float sh_q[32];
  __shared__ float sh_thetax[8*32];
  __shared__ float sh_thetay[8*32];
  __shared__ float sh_thetaz[8*32];

  // Process atoms pos to pos_end-1
  const unsigned int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    sh_q[threadIdx.x] = q;

    float w;

    w = x*recip11 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));
    w = y*recip22 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));
    w = z*recip33 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    sh_ix[threadIdx.x] = frxi;
    sh_iy[threadIdx.x] = fryi;
    sh_iz[threadIdx.x] = frzi;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float theta[8];

    calc_one_theta<float, 8>(wx, theta);
    sh_thetax[threadIdx.x*8 + 0] = theta[0];
    sh_thetax[threadIdx.x*8 + 1] = theta[1];
    sh_thetax[threadIdx.x*8 + 2] = theta[2];
    sh_thetax[threadIdx.x*8 + 3] = theta[3];
    sh_thetax[threadIdx.x*8 + 4] = theta[4];
    sh_thetax[threadIdx.x*8 + 5] = theta[5];
    sh_thetax[threadIdx.x*8 + 6] = theta[6];
    sh_thetax[threadIdx.x*8 + 7] = theta[7];

    calc_one_theta<float, 8>(wy, theta);
    sh_thetay[threadIdx.x*8 + 0] = theta[0];
    sh_thetay[threadIdx.x*8 + 1] = theta[1];
    sh_thetay[threadIdx.x*8 + 2] = theta[2];
    sh_thetay[threadIdx.x*8 + 3] = theta[3];
    sh_thetay[threadIdx.x*8 + 4] = theta[4];
    sh_thetay[threadIdx.x*8 + 5] = theta[5];
    sh_thetay[threadIdx.x*8 + 6] = theta[6];
    sh_thetay[threadIdx.x*8 + 7] = theta[7];

    calc_one_theta<float, 8>(wz, theta);
    sh_thetaz[threadIdx.x*8 + 0] = theta[0];
    sh_thetaz[threadIdx.x*8 + 1] = theta[1];
    sh_thetaz[threadIdx.x*8 + 2] = theta[2];
    sh_thetaz[threadIdx.x*8 + 3] = theta[3];
    sh_thetaz[threadIdx.x*8 + 4] = theta[4];
    sh_thetaz[threadIdx.x*8 + 5] = theta[5];
    sh_thetaz[threadIdx.x*8 + 6] = theta[6];
    sh_thetaz[threadIdx.x*8 + 7] = theta[7];

  }

  __syncthreads();

  // Grid point location, values of (ix0, iy0, iz0) are in range 0..7
  const int tid = (threadIdx.x + threadIdx.y*blockDim.x) % 512;   // 0...511
  const int x0 = tid % 8;
  const int y0 = (tid / 8) % 8;
  const int z0 = tid / 64;

  // Loop over atoms pos..pos_end-1
  int iadd = blockDim.x*blockDim.y/512;
  int i = (threadIdx.x + threadIdx.y*blockDim.x)/512;
  int iend = pos_end - blockIdx.x*blockDim.x;
  for (;i < iend;i += iadd) {
    int x = sh_ix[i] + x0;
    int y = sh_iy[i] + y0;
    int z = sh_iz[i] + z0;
    float q = sh_q[i];
      
    if (x >= nfftx) x -= nfftx;
    if (y >= nffty) y -= nffty;
    if (z >= nfftz) z -= nfftz;
      
    // Get position on the grid
    int ind = x + nfftx*(y + nffty*z);
      
    // Here we unroll the 8x8x8 loop with 512 threads.
    // Calculate interpolated charge value and store it to global memory
    write_grid<AT>(q*sh_thetax[i*8+x0]*sh_thetay[i*8+y0]*sh_thetaz[i*8+z0], ind, data);
  }

}

// Local structure for scalar_sum -function for energy and virial reductions
struct RecipVirial_t {
  double energy;
  double virial[6];
};

//
// Performs scalar sum on data(nfft1, nfft2, nfft3)
// T = float or double
// T2 = float2 or double2
//
template <typename T, typename T2, bool calc_energy_virial>
__global__ void scalar_sum_ortho_kernel(const int nfft1, const int nfft2, const int nfft3,
					const int size1, const int size2, const int size3,
					const int nf1, const int nf2, const int nf3,
					const T recip11, const T recip22, const T recip33,
					const T* prefac1, const T* prefac2, const T* prefac3,
					const T fac, const T piv_inv,
					const bool global_base, T2* data,
					double* __restrict__ energy_recip,
					Virial_t* __restrict__ virial) {
  extern __shared__ T sh_prefac[];

  // Create pointers to shared memory
  T* sh_prefac1 = (T *)&sh_prefac[0];
  T* sh_prefac2 = (T *)&sh_prefac[nfft1];
  T* sh_prefac3 = (T *)&sh_prefac[nfft1 + nfft2];

  // Calculate start position (k1, k2, k3) for each thread
  unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
  int k3 = tid/(size1*size2);
  tid -= k3*size1*size2;
  int k2 = tid/size1;
  int k1 = tid - k2*size1;

  // Calculate increments (k1_inc, k2_inc, k3_inc)
  int tot_inc = blockDim.x*gridDim.x;
  int k3_inc = tot_inc/(size1*size2);
  tot_inc -= k3_inc*size1*size2;
  int k2_inc = tot_inc/size1;
  int k1_inc = tot_inc - k2_inc*size1;

  // Set data[0] = 0 for the global (0,0,0)
  if (global_base && (blockIdx.x + threadIdx.x == 0)) {
    T2 zero;
    zero.x = (T)0;
    zero.y = (T)0;
    data[0] = zero;
    // Increment position
    k1 += k1_inc;
    if (k1 >= size1) {
      k1 -= size1;
      k2++;
    }
    k2 += k2_inc;
    if (k2 >= size2) {
      k2 -= size2;
      k3++;
    }
    k3 += k3_inc;
  }

  // Load prefac data into shared memory
  int pos = threadIdx.x;
  while (pos < nfft1) {
    sh_prefac1[pos] = prefac1[pos];
    pos += blockDim.x;
  }
  pos = threadIdx.x;
  while (pos < nfft2) {
    sh_prefac2[pos] = prefac2[pos];
    pos += blockDim.x;
  }
  pos = threadIdx.x;
  while (pos < nfft3) {
    sh_prefac3[pos] = prefac3[pos];
    pos += blockDim.x;
  }
  __syncthreads();

  double energy = 0.0;
  double virial0 = 0.0;
  double virial1 = 0.0;
  double virial2 = 0.0;
  double virial3 = 0.0;
  double virial4 = 0.0;
  double virial5 = 0.0;

  while (k3 < size3) {

    int pos = k1 + (k2 + k3*size2)*size1;
    T2 q = data[pos];

    int m1 = k1;
    int m2 = k2;
    int m3 = k3;
    if (k1 >= nf1) m1 -= nfft1;
    if (k2 >= nf2) m2 -= nfft2;
    if (k3 >= nf3) m3 -= nfft3;

    T mhat1 = recip11*m1;
    T mhat2 = recip22*m2;
    T mhat3 = recip33*m3;

    T msq = mhat1*mhat1 + mhat2*mhat2 + mhat3*mhat3;
    T msq_inv = (T)1.0/msq;

    // NOTE: check if it's faster to pre-calculate exp()
    T eterm = exp(-fac*msq)*piv_inv*sh_prefac1[k1]*sh_prefac2[k2]*sh_prefac3[k3]*msq_inv;

    if (calc_energy_virial) {
      T tmp1  = eterm*(q.x*q.x + q.y*q.y);
      T vterm  = ((T)2)*(fac + msq_inv);
      T tmp2   = tmp1*vterm;

      energy += (double)tmp1;
      virial0 += (double)(tmp1*(vterm*mhat1*mhat1 - ((T)1)));
      virial1 += (double)(tmp2*mhat1*mhat2);
      virial2 += (double)(tmp2*mhat1*mhat3);
      virial3 += (double)(tmp1*(vterm*mhat2*mhat2 - ((T)1)));
      virial4 += (double)(tmp2*mhat2*mhat3);
      virial5 += (double)(tmp1*(vterm*mhat3*mhat3 - ((T)1)));

      // The following is put into a separate if {} -block to avoid divergence within warp and
      // save registers
      if (k1 >= 1 && k1 < nfft1) {

	int k1s = nfft1 - (k1+1) + 1;
	int k2s = ((nfft2-(k2+1)+1) % nfft2);
	int k3s = ((nfft3-(k3+1)+1) % nfft3);

	int m1s = k1s;
	int m2s = k2s;
	int m3s = k3s;

	if (k1s >= nf1) m1s -= nfft1;
	if (k2s >= nf2) m2s -= nfft2;
	if (k3s >= nf3) m3s -= nfft3;

	T mhat1s = recip11*m1s;
	T mhat2s = recip22*m2s;
	T mhat3s = recip33*m3s;

	T msqs = mhat1s*mhat1s + mhat2s*mhat2s + mhat3s*mhat3s;
	T msqs_inv = ((T)1)/msqs;

	T eterms = exp(-fac*msqs)*piv_inv*sh_prefac1[k1s]*sh_prefac2[k2s]*sh_prefac3[k3s]*msqs_inv;

	T tmp1s  = eterms*(q.x*q.x + q.y*q.y);
	T vterms  = ((T)2)*(fac + msqs_inv);
	T tmp2s   = tmp1s*vterms;

	energy += (double)tmp1s;
	virial0 += (double)(tmp1s*(vterms*mhat1s*mhat1s - ((T)1)));
	virial1 += (double)(tmp2s*mhat1s*mhat2s);
	virial2 += (double)(tmp2s*mhat1s*mhat3s);
	virial3 += (double)(tmp1s*(vterms*mhat2s*mhat2s - ((T)1)));
	virial4 += (double)(tmp2s*mhat2s*mhat3s);
	virial5 += (double)(tmp1s*(vterms*mhat3s*mhat3s - ((T)1)));
      }
    }

    q.x *= eterm;
    q.y *= eterm;
    data[pos] = q;
    
    // Increment position
    k1 += k1_inc;
    if (k1 >= size1) {
      k1 -= size1;
      k2++;
    }
    k2 += k2_inc;
    if (k2 >= size2) {
      k2 -= size2;
      k3++;
    }
    k3 += k3_inc;
  }

  // Reduce energy and virial
  if (calc_energy_virial) {
#if __CUDA_ARCH__ < 300
    // Requires blockDim.x*sizeof(RecipVirial_t) amount of shared memory
    volatile RecipVirial_t* sh_ev = (RecipVirial_t *)sh_prefac;
    // NOTE: this __syncthreads() is needed because we're using a single shared memory buffer
    __syncthreads();
    sh_ev[threadIdx.x].energy  = energy;
    sh_ev[threadIdx.x].virial[0] = virial0;
    sh_ev[threadIdx.x].virial[1] = virial1;
    sh_ev[threadIdx.x].virial[2] = virial2;
    sh_ev[threadIdx.x].virial[3] = virial3;
    sh_ev[threadIdx.x].virial[4] = virial4;
    sh_ev[threadIdx.x].virial[5] = virial5;
    __syncthreads();
#endif
#if __CUDA_ARCH__ < 300
    for (int d=1;d < blockDim.x;d *= 2) {
      int t = threadIdx.x + d;
      double energy_val = (t < blockDim.x) ? sh_ev[t].energy : 0.0;
      double virial0_val = (t < blockDim.x) ? sh_ev[t].virial[0] : 0.0;
      double virial1_val = (t < blockDim.x) ? sh_ev[t].virial[1] : 0.0;
      double virial2_val = (t < blockDim.x) ? sh_ev[t].virial[2] : 0.0;
      double virial3_val = (t < blockDim.x) ? sh_ev[t].virial[3] : 0.0;
      double virial4_val = (t < blockDim.x) ? sh_ev[t].virial[4] : 0.0;
      double virial5_val = (t < blockDim.x) ? sh_ev[t].virial[5] : 0.0;
      __syncthreads();
      sh_ev[threadIdx.x].energy += energy_val;
      sh_ev[threadIdx.x].virial[0] += virial0_val;
      sh_ev[threadIdx.x].virial[1] += virial1_val;
      sh_ev[threadIdx.x].virial[2] += virial2_val;
      sh_ev[threadIdx.x].virial[3] += virial3_val;
      sh_ev[threadIdx.x].virial[4] += virial4_val;
      sh_ev[threadIdx.x].virial[5] += virial5_val;
      __syncthreads();
    }
#else
    const int tid = threadIdx.x & (warpsize-1);
    const int base = (threadIdx.x/warpsize);
    volatile RecipVirial_t* sh_ev = (RecipVirial_t *)sh_prefac;
    // Reduce within warps
    for (int d=warpsize/2;d >= 1;d /= 2) {
      energy += __hiloint2double(__shfl(__double2hiint(energy), tid+d),
				 __shfl(__double2loint(energy), tid+d));
      virial0 += __hiloint2double(__shfl(__double2hiint(virial0), tid+d),
				  __shfl(__double2loint(virial0), tid+d));
      virial1 += __hiloint2double(__shfl(__double2hiint(virial1), tid+d),
				  __shfl(__double2loint(virial1), tid+d));
      virial2 += __hiloint2double(__shfl(__double2hiint(virial2), tid+d),
				  __shfl(__double2loint(virial2), tid+d));
      virial3 += __hiloint2double(__shfl(__double2hiint(virial3), tid+d),
				  __shfl(__double2loint(virial3), tid+d));
      virial4 += __hiloint2double(__shfl(__double2hiint(virial4), tid+d),
				  __shfl(__double2loint(virial4), tid+d));
      virial5 += __hiloint2double(__shfl(__double2hiint(virial5), tid+d),
				  __shfl(__double2loint(virial5), tid+d));
    }
    // Reduce between warps
    // NOTE: this __syncthreads() is needed because we're using a single shared memory buffer
    __syncthreads();
    if (tid == 0) {
      sh_ev[base].energy = energy;
      sh_ev[base].virial[0] = virial0;
      sh_ev[base].virial[1] = virial1;
      sh_ev[base].virial[2] = virial2;
      sh_ev[base].virial[3] = virial3;
      sh_ev[base].virial[4] = virial4;
      sh_ev[base].virial[5] = virial5;
    }
    __syncthreads();
    if (base == 0) {
      energy = (tid < blockDim.x/warpsize) ? sh_ev[tid].energy : 0.0;
      virial0 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[0] : 0.0;
      virial1 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[1] : 0.0;
      virial2 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[2] : 0.0;
      virial3 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[3] : 0.0;
      virial4 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[4] : 0.0;
      virial5 = (tid < blockDim.x/warpsize) ? sh_ev[tid].virial[5] : 0.0;
      for (int d=warpsize/2;d >= 1;d /= 2) {
	energy += __hiloint2double(__shfl(__double2hiint(energy), tid+d),
				   __shfl(__double2loint(energy), tid+d));
	virial0 += __hiloint2double(__shfl(__double2hiint(virial0), tid+d),
				    __shfl(__double2loint(virial0), tid+d));
	virial1 += __hiloint2double(__shfl(__double2hiint(virial1), tid+d),
				    __shfl(__double2loint(virial1), tid+d));
	virial2 += __hiloint2double(__shfl(__double2hiint(virial2), tid+d),
				    __shfl(__double2loint(virial2), tid+d));
	virial3 += __hiloint2double(__shfl(__double2hiint(virial3), tid+d),
				    __shfl(__double2loint(virial3), tid+d));
	virial4 += __hiloint2double(__shfl(__double2hiint(virial4), tid+d),
				    __shfl(__double2loint(virial4), tid+d));
	virial5 += __hiloint2double(__shfl(__double2hiint(virial5), tid+d),
				    __shfl(__double2loint(virial5), tid+d));
      }
    }
    
#endif

    if (threadIdx.x == 0) {
#if __CUDA_ARCH__ < 300
      energy = sh_ev[0].energy;
      virial0 = sh_ev[0].virial[0];
      virial1 = sh_ev[0].virial[1];
      virial2 = sh_ev[0].virial[2];
      virial3 = sh_ev[0].virial[3];
      virial4 = sh_ev[0].virial[4];
      virial5 = sh_ev[0].virial[5];
#endif
      atomicAdd(energy_recip, energy*half_ccelec);
      virial0 *= -half_ccelec;
      virial1 *= -half_ccelec;
      virial2 *= -half_ccelec;
      virial3 *= -half_ccelec;
      virial4 *= -half_ccelec;
      virial5 *= -half_ccelec;
      atomicAdd(&virial->virmat[0], virial0);
      atomicAdd(&virial->virmat[1], virial1);
      atomicAdd(&virial->virmat[2], virial2);
      atomicAdd(&virial->virmat[3], virial1);
      atomicAdd(&virial->virmat[4], virial3);
      atomicAdd(&virial->virmat[5], virial4);
      atomicAdd(&virial->virmat[6], virial2);
      atomicAdd(&virial->virmat[7], virial4);
      atomicAdd(&virial->virmat[8], virial5);
    }

  }

  /*
  // Set data[0] = 0 for the global (0,0,0)
  if (global_base && (blockIdx.x + threadIdx.x == 0)) {
    T2 zero;
    zero.x = (T)0;
    zero.y = (T)0;
    data[0] = zero;
  }
  */

}

#ifndef USE_TEXTURE_OBJECTS
texture<float, 1, hipReadModeElementType> gridTexRef;
#endif

// Per atom data structure for the gather_force -kernels
template <typename T, int order>
struct gather_t {
  int ix;
  int iy;
  int iz;
  T charge;
  T thetax[order];
  T thetay[order];
  T thetaz[order];
  T dthetax[order];
  T dthetay[order];
  T dthetaz[order];
  float f1;
  float f2;
  float f3;
};

template <typename T>
__forceinline__ __device__ void write_force_atomic(const float fx,
						   const float fy,
						   const float fz,
						   const int ind,
						   const int stride,
						   const int stride2,
						   T* force) {
  // The generic version can not be used for anything
}

template <typename T>
__forceinline__ __device__ void write_force(const float fx,
					    const float fy,
					    const float fz,
					    const int ind,
					    const int stride,
					    const int stride2,
					    T* force) {
  // The generic version can not be used for anything
}

// Template specialization for 64bit integer = "long long int"
template <>
__forceinline__ __device__ void write_force_atomic <long long int> (const float fx,
								    const float fy,
								    const float fz,
								    const int ind,
								    const int stride,
								    const int stride2,
								    long long int* force) {
  unsigned long long int fx_ulli = llitoulli(lliroundf(FORCE_SCALE*fx));
  unsigned long long int fy_ulli = llitoulli(lliroundf(FORCE_SCALE*fy));
  unsigned long long int fz_ulli = llitoulli(lliroundf(FORCE_SCALE*fz));
  atomicAdd((unsigned long long int *)&force[ind          ], fx_ulli);
  atomicAdd((unsigned long long int *)&force[ind + stride ], fy_ulli);
  atomicAdd((unsigned long long int *)&force[ind + stride2], fz_ulli);
}

// Template specialization for 64bit integer = "long long int"
template <>
__forceinline__ __device__ void write_force <long long int> (const float fx,
							     const float fy,
							     const float fz,
							     const int ind,
							     const int stride,
							     const int stride2,
							     long long int* force) {
  unsigned long long int fx_ulli = llitoulli(lliroundf(FORCE_SCALE*fx));
  unsigned long long int fy_ulli = llitoulli(lliroundf(FORCE_SCALE*fy));
  unsigned long long int fz_ulli = llitoulli(lliroundf(FORCE_SCALE*fz));
  unsigned long long int *force_ulli = (unsigned long long int *)force;
  force_ulli[ind          ] += fx_ulli;
  force_ulli[ind + stride ] += fy_ulli;
  force_ulli[ind + stride2] += fz_ulli;
}


//-----------------------------------------------------------------------------------------
// Generic version can not be used
template <typename T> __forceinline__ __device__
void gather_force_store(const float fx, const float fy, const float fz,
			const int stride, const int pos,
			T* force) {
}

// Template specialization for "long long int"
template <> __forceinline__ __device__
void gather_force_store<long long int>(const float fx, const float fy, const float fz, 
				       const int stride, const int pos, 
				       long long int* force) {
  // Add into strided "long long int" array
  long long int fx_lli = lliroundf(fx*FORCE_SCALE);
  long long int fy_lli = lliroundf(fy*FORCE_SCALE);
  long long int fz_lli = lliroundf(fz*FORCE_SCALE);
  write_force<long long int>(fx_lli, fy_lli, fz_lli, pos, stride, force);
}

// Template specialization for "float"
template <> __forceinline__ __device__
void gather_force_store<float>(const float fx, const float fy, const float fz, 
			       const int stride, const int pos, 
			       float* force) {
  // Store into non-strided float XYZ array
  force[pos]          = fx;
  force[pos+stride]   = fy;
  force[pos+stride*2] = fz;
}

// Template specialization for "float3"
template <> __forceinline__ __device__
void gather_force_store<float3>(const float fx, const float fy, const float fz, 
				const int stride, const int pos, 
				float3* force) {
  // Store into non-strided "float3" array
  force[pos].x = fx;
  force[pos].y = fy;
  force[pos].z = fz;
}
//-----------------------------------------------------------------------------------------

//
// Gathers forces from the grid
// blockDim.x            = Number of atoms each block loads
// blockDim.x*blockDim.y = Total number of threads per block
//
template <typename CT>
__global__ void gather_force_4_ortho_kernel(const int ncoord,
					    const int nfftx, const int nffty, const int nfftz,
					    const int xsize, const int ysize, const int zsize,
					    const float recip1, const float recip2, const float recip3,
					    const int *gix, const int *giy, const int *giz,
					    const float *charge,
					    const float4 *thetax, const float4 *thetay,
					    const float4 *thetaz,
					    const float4 *dthetax, const float4 *dthetay,
					    const float4 *dthetaz,
#ifdef USE_TEXTURE_OBJECTS
					    const hipTextureObject_t gridTexObj,
#endif
					    const int stride,
					    CT *force) {
  // Shared memory
  extern __shared__ gather_t<CT, 4> shbuf[];

  const int tid = threadIdx.x + threadIdx.y*blockDim.x;
  volatile gather_t<CT, 4> *shmem = shbuf;
  volatile float3 *shred = &((float3 *)&shbuf[blockDim.x])[(tid/8)*8];

  const int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end && threadIdx.y == 0) {
    shmem[threadIdx.x].ix = gix[pos];
    shmem[threadIdx.x].iy = giy[pos];
    shmem[threadIdx.x].iz = giz[pos];
    shmem[threadIdx.x].charge = charge[pos];

    float4 tmpx = thetax[pos];
    float4 tmpy = thetay[pos];
    float4 tmpz = thetaz[pos];

    shmem[threadIdx.x].thetax[0] = tmpx.x;
    shmem[threadIdx.x].thetax[1] = tmpx.y;
    shmem[threadIdx.x].thetax[2] = tmpx.z;
    shmem[threadIdx.x].thetax[3] = tmpx.w;

    shmem[threadIdx.x].thetay[0] = tmpy.x;
    shmem[threadIdx.x].thetay[1] = tmpy.y;
    shmem[threadIdx.x].thetay[2] = tmpy.z;
    shmem[threadIdx.x].thetay[3] = tmpy.w;

    shmem[threadIdx.x].thetaz[0] = tmpz.x;
    shmem[threadIdx.x].thetaz[1] = tmpz.y;
    shmem[threadIdx.x].thetaz[2] = tmpz.z;
    shmem[threadIdx.x].thetaz[3] = tmpz.w;

    tmpx = dthetax[pos];
    tmpy = dthetay[pos];
    tmpz = dthetaz[pos];

    shmem[threadIdx.x].dthetax[0] = tmpx.x;
    shmem[threadIdx.x].dthetax[1] = tmpx.y;
    shmem[threadIdx.x].dthetax[2] = tmpx.z;
    shmem[threadIdx.x].dthetax[3] = tmpx.w;

    shmem[threadIdx.x].dthetay[0] = tmpy.x;
    shmem[threadIdx.x].dthetay[1] = tmpy.y;
    shmem[threadIdx.x].dthetay[2] = tmpy.z;
    shmem[threadIdx.x].dthetay[3] = tmpy.w;

    shmem[threadIdx.x].dthetaz[0] = tmpz.x;
    shmem[threadIdx.x].dthetaz[1] = tmpz.y;
    shmem[threadIdx.x].dthetaz[2] = tmpz.z;
    shmem[threadIdx.x].dthetaz[3] = tmpz.w;

  }
  __syncthreads();

  // Calculate the index this thread is calculating
  const int tx = 0;             // 0
  const int ty = (tid & 1);     // 0, 1
  const int tz = (tid/2) & 3;   // 0, 1, 2, 3

  // Calculate force by looping 64/8=8 times
  int base = tid/8;
  const int base_end = pos_end - blockIdx.x*blockDim.x;
  while (base < base_end) {
    int ix0 = shmem[base].ix + tx;
    int iy0 = shmem[base].iy + ty;
    int iz0 = shmem[base].iz + tz;

    int ix1 = ix0 + 1;
    int ix2 = ix0 + 2;
    int ix3 = ix0 + 3;

    int iy1 = iy0 + 2;

    if (ix0 >= nfftx) ix0 -= nfftx;
    if (iy0 >= nffty) iy0 -= nffty;
    if (iz0 >= nfftz) iz0 -= nfftz;

    if (ix1 >= nfftx) ix1 -= nfftx;
    if (ix2 >= nfftx) ix2 -= nfftx;
    if (ix3 >= nfftx) ix3 -= nfftx;

    if (iy1 >= nffty) iy1 -= nffty;

#ifdef USE_TEXTURE_OBJECTS
    float q0 = tex1Dfetch<float>(gridTexObj, ix0 + (iy0 + iz0*ysize)*xsize);
    float q1 = tex1Dfetch<float>(gridTexObj, ix1 + (iy0 + iz0*ysize)*xsize);
    float q2 = tex1Dfetch<float>(gridTexObj, ix2 + (iy0 + iz0*ysize)*xsize);
    float q3 = tex1Dfetch<float>(gridTexObj, ix3 + (iy0 + iz0*ysize)*xsize);
    float q4 = tex1Dfetch<float>(gridTexObj, ix0 + (iy1 + iz0*ysize)*xsize);
    float q5 = tex1Dfetch<float>(gridTexObj, ix1 + (iy1 + iz0*ysize)*xsize);
    float q6 = tex1Dfetch<float>(gridTexObj, ix2 + (iy1 + iz0*ysize)*xsize);
    float q7 = tex1Dfetch<float>(gridTexObj, ix3 + (iy1 + iz0*ysize)*xsize);
#else
    float q0 = tex1Dfetch(gridTexRef, ix0 + (iy0 + iz0*ysize)*xsize);
    float q1 = tex1Dfetch(gridTexRef, ix1 + (iy0 + iz0*ysize)*xsize);
    float q2 = tex1Dfetch(gridTexRef, ix2 + (iy0 + iz0*ysize)*xsize);
    float q3 = tex1Dfetch(gridTexRef, ix3 + (iy0 + iz0*ysize)*xsize);
    float q4 = tex1Dfetch(gridTexRef, ix0 + (iy1 + iz0*ysize)*xsize);
    float q5 = tex1Dfetch(gridTexRef, ix1 + (iy1 + iz0*ysize)*xsize);
    float q6 = tex1Dfetch(gridTexRef, ix2 + (iy1 + iz0*ysize)*xsize);
    float q7 = tex1Dfetch(gridTexRef, ix3 + (iy1 + iz0*ysize)*xsize);
#endif
    
    float thx0 = shmem[base].thetax[tx+0];
    float thx1 = shmem[base].thetax[tx+1];
    float thx2 = shmem[base].thetax[tx+2];
    float thx3 = shmem[base].thetax[tx+3];
    float thy0 = shmem[base].thetay[ty];
    float thy1 = shmem[base].thetay[ty+2];
    float thz0 = shmem[base].thetaz[tz];

    float dthx0 = shmem[base].dthetax[tx+0];
    float dthx1 = shmem[base].dthetax[tx+1];
    float dthx2 = shmem[base].dthetax[tx+2];
    float dthx3 = shmem[base].dthetax[tx+3];
    float dthy0 = shmem[base].dthetay[ty];
    float dthy1 = shmem[base].dthetay[ty+2];
    float dthz0 = shmem[base].dthetaz[tz];

    float thy0_thz0  = thy0 * thz0;
    float dthy0_thz0 = dthy0 * thz0;
    float thy0_dthz0 = thy0 * dthz0;

    float thy1_thz0  = thy1 * thz0;
    float dthy1_thz0 = dthy1 * thz0;
    float thy1_dthz0 = thy1 * dthz0;

    float f1 = dthx0 * thy0_thz0 * q0;
    float f2 = thx0 * dthy0_thz0 * q0;
    float f3 = thx0 * thy0_dthz0 * q0;

    f1 += dthx1 * thy0_thz0 * q1;
    f2 += thx1 * dthy0_thz0 * q1;
    f3 += thx1 * thy0_dthz0 * q1;

    f1 += dthx2 * thy0_thz0 * q2;
    f2 += thx2 * dthy0_thz0 * q2;
    f3 += thx2 * thy0_dthz0 * q2;

    f1 += dthx3 * thy0_thz0 * q3;
    f2 += thx3 * dthy0_thz0 * q3;
    f3 += thx3 * thy0_dthz0 * q3;

    f1 += dthx0 * thy1_thz0 * q4;
    f2 += thx0 * dthy1_thz0 * q4;
    f3 += thx0 * thy1_dthz0 * q4;

    f1 += dthx1 * thy1_thz0 * q5;
    f2 += thx1 * dthy1_thz0 * q5;
    f3 += thx1 * thy1_dthz0 * q5;

    f1 += dthx2 * thy1_thz0 * q6;
    f2 += thx2 * dthy1_thz0 * q6;
    f3 += thx2 * thy1_dthz0 * q6;

    f1 += dthx3 * thy1_thz0 * q7;
    f2 += thx3 * dthy1_thz0 * q7;
    f3 += thx3 * thy1_dthz0 * q7;

    // Reduce
    const int i = threadIdx.x & 7;
    shred[i].x = f1;
    shred[i].y = f2;
    shred[i].z = f3;

    if (i < 4) {
      shred[i].x += shred[i+4].x;
      shred[i].y += shred[i+4].y;
      shred[i].z += shred[i+4].z;
    }

    if (i < 2) {
      shred[i].x += shred[i+2].x;
      shred[i].y += shred[i+2].y;
      shred[i].z += shred[i+2].z;
    }

    if (i == 0) {
      shmem[base].f1 = shred[0].x + shred[1].x;
      shmem[base].f2 = shred[0].y + shred[1].y;
      shmem[base].f3 = shred[0].z + shred[1].z;
    }

    base += 8;
  }

  // Write forces
  const int stride2 = 2*stride;
  __syncthreads();
  if (pos < pos_end) {
    float f1 = shmem[threadIdx.x].f1;
    float f2 = shmem[threadIdx.x].f2;
    float f3 = shmem[threadIdx.x].f3;
    float q = shmem[threadIdx.x].charge;
    float fx = q*recip1*f1;
    float fy = q*recip2*f2;
    float fz = q*recip3*f3;
    force[pos]         = fx;
    force[pos+stride]  = fy;
    force[pos+stride2] = fz;
  }

}

//
// Gathers forces from the grid
// blockDim.x            = Number of atoms each block loads
// blockDim.x*blockDim.y = Total number of threads per block
//
template <typename CT, typename FT>
__global__ void gather_force_4_ortho_kernel(const float4 *xyzq, const int ncoord,
					    const int nfftx, const int nffty, const int nfftz,
					    const int xsize, const int ysize, const int zsize,
					    const float recip1, const float recip2, const float recip3,
					    const float ccelec,
#ifdef USE_TEXTURE_OBJECTS
					    const hipTextureObject_t gridTexObj,
#endif
					    const int stride,
					    FT *force) {

  const int tid = threadIdx.x + threadIdx.y*blockDim.x; // 0...63

  // Shared memory
  __shared__ gather_t<CT, 4> shmem[32];
#if __CUDA_ARCH__ < 300
  __shared__ float3 shred_buf[32*2];
  volatile float3 *shred = &shred_buf[(tid/8)*8];
#endif

  //  extern __shared__ gather_t<CT> shbuf[];

  /*
  volatile gather_t<CT> *shmem = shbuf;
#if __CUDA_ARCH__ < 300
  volatile float3 *shred = &((float3 *)&shbuf[blockDim.x])[(tid/8)*8];
#endif
  */

  const int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    float w;

    w = x*recip1 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));

    w = y*recip2 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));

    w = z*recip3 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    shmem[threadIdx.x].ix = frxi;
    shmem[threadIdx.x].iy = fryi;
    shmem[threadIdx.x].iz = frzi;
    shmem[threadIdx.x].charge = q;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float3 theta_tmp[4];
    float3 dtheta_tmp[4];
    calc_theta_dtheta_4<float, float3>(wx, wy, wz, theta_tmp, dtheta_tmp);
    
    shmem[threadIdx.x].thetax[0] = theta_tmp[0].x;
    shmem[threadIdx.x].thetax[1] = theta_tmp[1].x;
    shmem[threadIdx.x].thetax[2] = theta_tmp[2].x;
    shmem[threadIdx.x].thetax[3] = theta_tmp[3].x;

    shmem[threadIdx.x].thetay[0] = theta_tmp[0].y;
    shmem[threadIdx.x].thetay[1] = theta_tmp[1].y;
    shmem[threadIdx.x].thetay[2] = theta_tmp[2].y;
    shmem[threadIdx.x].thetay[3] = theta_tmp[3].y;

    shmem[threadIdx.x].thetaz[0] = theta_tmp[0].z;
    shmem[threadIdx.x].thetaz[1] = theta_tmp[1].z;
    shmem[threadIdx.x].thetaz[2] = theta_tmp[2].z;
    shmem[threadIdx.x].thetaz[3] = theta_tmp[3].z;

    shmem[threadIdx.x].dthetax[0] = dtheta_tmp[0].x;
    shmem[threadIdx.x].dthetax[1] = dtheta_tmp[1].x;
    shmem[threadIdx.x].dthetax[2] = dtheta_tmp[2].x;
    shmem[threadIdx.x].dthetax[3] = dtheta_tmp[3].x;

    shmem[threadIdx.x].dthetay[0] = dtheta_tmp[0].y;
    shmem[threadIdx.x].dthetay[1] = dtheta_tmp[1].y;
    shmem[threadIdx.x].dthetay[2] = dtheta_tmp[2].y;
    shmem[threadIdx.x].dthetay[3] = dtheta_tmp[3].y;

    shmem[threadIdx.x].dthetaz[0] = dtheta_tmp[0].z;
    shmem[threadIdx.x].dthetaz[1] = dtheta_tmp[1].z;
    shmem[threadIdx.x].dthetaz[2] = dtheta_tmp[2].z;
    shmem[threadIdx.x].dthetaz[3] = dtheta_tmp[3].z;

  }
  __syncthreads();

  // Calculate the index this thread is calculating
  const int tx = 0;             // 0
  const int ty = (tid & 1);     // 0, 1
  const int tz = (tid/2) & 3;   // 0, 1, 2, 3

  //
  // Calculate forces for 32 atoms. We have 32*2 = 64 threads
  // Loop is iterated 4 times:
  //                         (iteration)
  // Threads 0...7   = atoms 0, 8,  16, 24
  // Threads 8...15  = atoms 1, 9,  17, 25
  // Threads 16...31 = atoms 2, 10, 18, 26
  //                ...
  // Threads 56...63 = atoms 7, 15, 23, 31
  //
  int base = tid/8;
  const int base_end = pos_end - blockIdx.x*blockDim.x;
  while (base < base_end) {
    int ix0 = shmem[base].ix + tx;
    int iy0 = shmem[base].iy + ty;
    int iz0 = shmem[base].iz + tz;

    int ix1 = ix0 + 1;
    int ix2 = ix0 + 2;
    int ix3 = ix0 + 3;

    int iy1 = iy0 + 2;

    if (ix0 >= nfftx) ix0 -= nfftx;
    if (iy0 >= nffty) iy0 -= nffty;
    if (iz0 >= nfftz) iz0 -= nfftz;

    if (ix1 >= nfftx) ix1 -= nfftx;
    if (ix2 >= nfftx) ix2 -= nfftx;
    if (ix3 >= nfftx) ix3 -= nfftx;

    if (iy1 >= nffty) iy1 -= nffty;

#ifdef USE_TEXTURE_OBJECTS
    float q0 = tex1Dfetch<float>(gridTexObj, ix0 + (iy0 + iz0*ysize)*xsize);
    float q1 = tex1Dfetch<float>(gridTexObj, ix1 + (iy0 + iz0*ysize)*xsize);
    float q2 = tex1Dfetch<float>(gridTexObj, ix2 + (iy0 + iz0*ysize)*xsize);
    float q3 = tex1Dfetch<float>(gridTexObj, ix3 + (iy0 + iz0*ysize)*xsize);
    float q4 = tex1Dfetch<float>(gridTexObj, ix0 + (iy1 + iz0*ysize)*xsize);
    float q5 = tex1Dfetch<float>(gridTexObj, ix1 + (iy1 + iz0*ysize)*xsize);
    float q6 = tex1Dfetch<float>(gridTexObj, ix2 + (iy1 + iz0*ysize)*xsize);
    float q7 = tex1Dfetch<float>(gridTexObj, ix3 + (iy1 + iz0*ysize)*xsize);
#else
    float q0 = tex1Dfetch(gridTexRef, ix0 + (iy0 + iz0*ysize)*xsize);
    float q1 = tex1Dfetch(gridTexRef, ix1 + (iy0 + iz0*ysize)*xsize);
    float q2 = tex1Dfetch(gridTexRef, ix2 + (iy0 + iz0*ysize)*xsize);
    float q3 = tex1Dfetch(gridTexRef, ix3 + (iy0 + iz0*ysize)*xsize);
    float q4 = tex1Dfetch(gridTexRef, ix0 + (iy1 + iz0*ysize)*xsize);
    float q5 = tex1Dfetch(gridTexRef, ix1 + (iy1 + iz0*ysize)*xsize);
    float q6 = tex1Dfetch(gridTexRef, ix2 + (iy1 + iz0*ysize)*xsize);
    float q7 = tex1Dfetch(gridTexRef, ix3 + (iy1 + iz0*ysize)*xsize);
#endif
    
    float thx0 = shmem[base].thetax[tx+0];
    float thx1 = shmem[base].thetax[tx+1];
    float thx2 = shmem[base].thetax[tx+2];
    float thx3 = shmem[base].thetax[tx+3];
    float thy0 = shmem[base].thetay[ty];
    float thy1 = shmem[base].thetay[ty+2];
    float thz0 = shmem[base].thetaz[tz];

    float dthx0 = shmem[base].dthetax[tx+0];
    float dthx1 = shmem[base].dthetax[tx+1];
    float dthx2 = shmem[base].dthetax[tx+2];
    float dthx3 = shmem[base].dthetax[tx+3];
    float dthy0 = shmem[base].dthetay[ty];
    float dthy1 = shmem[base].dthetay[ty+2];
    float dthz0 = shmem[base].dthetaz[tz];

    float thy0_thz0  = thy0 * thz0;
    float dthy0_thz0 = dthy0 * thz0;
    float thy0_dthz0 = thy0 * dthz0;

    float thy1_thz0  = thy1 * thz0;
    float dthy1_thz0 = dthy1 * thz0;
    float thy1_dthz0 = thy1 * dthz0;

    float f1 = dthx0 * thy0_thz0 * q0;
    float f2 = thx0 * dthy0_thz0 * q0;
    float f3 = thx0 * thy0_dthz0 * q0;

    f1 += dthx1 * thy0_thz0 * q1;
    f2 += thx1 * dthy0_thz0 * q1;
    f3 += thx1 * thy0_dthz0 * q1;

    f1 += dthx2 * thy0_thz0 * q2;
    f2 += thx2 * dthy0_thz0 * q2;
    f3 += thx2 * thy0_dthz0 * q2;

    f1 += dthx3 * thy0_thz0 * q3;
    f2 += thx3 * dthy0_thz0 * q3;
    f3 += thx3 * thy0_dthz0 * q3;

    f1 += dthx0 * thy1_thz0 * q4;
    f2 += thx0 * dthy1_thz0 * q4;
    f3 += thx0 * thy1_dthz0 * q4;

    f1 += dthx1 * thy1_thz0 * q5;
    f2 += thx1 * dthy1_thz0 * q5;
    f3 += thx1 * thy1_dthz0 * q5;

    f1 += dthx2 * thy1_thz0 * q6;
    f2 += thx2 * dthy1_thz0 * q6;
    f3 += thx2 * thy1_dthz0 * q6;

    f1 += dthx3 * thy1_thz0 * q7;
    f2 += thx3 * dthy1_thz0 * q7;
    f3 += thx3 * thy1_dthz0 * q7;

    // Reduce
#if __CUDA_ARCH__ >= 300
    const int i = threadIdx.x & 7;

    f1 += __shfl(f1, i+4, 8);
    f2 += __shfl(f2, i+4, 8);
    f3 += __shfl(f3, i+4, 8);

    f1 += __shfl(f1, i+2, 8);
    f2 += __shfl(f2, i+2, 8);
    f3 += __shfl(f3, i+2, 8);

    f1 += __shfl(f1, i+1, 8);
    f2 += __shfl(f2, i+1, 8);
    f3 += __shfl(f3, i+1, 8);

    if (i == 0) {
      shmem[base].f1 = f1;
      shmem[base].f2 = f2;
      shmem[base].f3 = f3;
    }

#else
    const int i = threadIdx.x & 7;
    shred[i].x = f1;
    shred[i].y = f2;
    shred[i].z = f3;

    if (i < 4) {
      shred[i].x += shred[i+4].x;
      shred[i].y += shred[i+4].y;
      shred[i].z += shred[i+4].z;
    }

    if (i < 2) {
      shred[i].x += shred[i+2].x;
      shred[i].y += shred[i+2].y;
      shred[i].z += shred[i+2].z;
    }

    if (i == 0) {
      shmem[base].f1 = shred[0].x + shred[1].x;
      shmem[base].f2 = shred[0].y + shred[1].y;
      shmem[base].f3 = shred[0].z + shred[1].z;
    }
#endif

    base += 8;
  }

  // Write forces
  __syncthreads();
  if (pos < pos_end && threadIdx.y == 0) {
    float f1 = shmem[threadIdx.x].f1;
    float f2 = shmem[threadIdx.x].f2;
    float f3 = shmem[threadIdx.x].f3;
    float q = shmem[threadIdx.x].charge*ccelec;
    float fx = q*recip1*f1*nfftx;
    float fy = q*recip2*f2*nffty;
    float fz = q*recip3*f3*nfftz;
    gather_force_store<FT>(fx, fy, fz, stride, pos, force);
  }

}

//
// Gathers forces from the grid
// blockDim.x            = Number of atoms each block loads
// blockDim.x*blockDim.y = Total number of threads per block
//
template <typename CT, typename FT>
__global__ void gather_force_6_ortho_kernel(const float4 *xyzq, const int ncoord,
					    const int nfftx, const int nffty, const int nfftz,
					    const int xsize, const int ysize, const int zsize,
					    const float recip1, const float recip2, const float recip3,
					    const float ccelec,
#ifdef USE_TEXTURE_OBJECTS
					    const hipTextureObject_t gridTexObj,
#endif
					    const int stride,
					    FT *force) {

  const int tid = threadIdx.x + threadIdx.y*blockDim.x; // 0...63

  // Shared memory
  __shared__ gather_t<CT, 6> shmem[32];
#if __CUDA_ARCH__ < 300
  __shared__ float3 shred_buf[32*2];
  volatile float3 *shred = &shred_buf[(tid/8)*8];
#endif

  const int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    float w;

    w = x*recip1 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));

    w = y*recip2 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));

    w = z*recip3 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    shmem[threadIdx.x].ix = frxi;
    shmem[threadIdx.x].iy = fryi;
    shmem[threadIdx.x].iz = frzi;
    shmem[threadIdx.x].charge = q;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float3 theta_tmp[6];
    float3 dtheta_tmp[6];
    calc_theta_dtheta<float, float3, 6>(wx, wy, wz, theta_tmp, dtheta_tmp);
    
    shmem[threadIdx.x].thetax[0] = theta_tmp[0].x;
    shmem[threadIdx.x].thetax[1] = theta_tmp[1].x;
    shmem[threadIdx.x].thetax[2] = theta_tmp[2].x;
    shmem[threadIdx.x].thetax[3] = theta_tmp[3].x;
    shmem[threadIdx.x].thetax[4] = theta_tmp[4].x;
    shmem[threadIdx.x].thetax[5] = theta_tmp[5].x;

    shmem[threadIdx.x].thetay[0] = theta_tmp[0].y;
    shmem[threadIdx.x].thetay[1] = theta_tmp[1].y;
    shmem[threadIdx.x].thetay[2] = theta_tmp[2].y;
    shmem[threadIdx.x].thetay[3] = theta_tmp[3].y;
    shmem[threadIdx.x].thetay[4] = theta_tmp[4].y;
    shmem[threadIdx.x].thetay[5] = theta_tmp[5].y;

    shmem[threadIdx.x].thetaz[0] = theta_tmp[0].z;
    shmem[threadIdx.x].thetaz[1] = theta_tmp[1].z;
    shmem[threadIdx.x].thetaz[2] = theta_tmp[2].z;
    shmem[threadIdx.x].thetaz[3] = theta_tmp[3].z;
    shmem[threadIdx.x].thetaz[4] = theta_tmp[4].z;
    shmem[threadIdx.x].thetaz[5] = theta_tmp[5].z;

    shmem[threadIdx.x].dthetax[0] = dtheta_tmp[0].x;
    shmem[threadIdx.x].dthetax[1] = dtheta_tmp[1].x;
    shmem[threadIdx.x].dthetax[2] = dtheta_tmp[2].x;
    shmem[threadIdx.x].dthetax[3] = dtheta_tmp[3].x;
    shmem[threadIdx.x].dthetax[4] = dtheta_tmp[4].x;
    shmem[threadIdx.x].dthetax[5] = dtheta_tmp[5].x;

    shmem[threadIdx.x].dthetay[0] = dtheta_tmp[0].y;
    shmem[threadIdx.x].dthetay[1] = dtheta_tmp[1].y;
    shmem[threadIdx.x].dthetay[2] = dtheta_tmp[2].y;
    shmem[threadIdx.x].dthetay[3] = dtheta_tmp[3].y;
    shmem[threadIdx.x].dthetay[4] = dtheta_tmp[4].y;
    shmem[threadIdx.x].dthetay[5] = dtheta_tmp[5].y;

    shmem[threadIdx.x].dthetaz[0] = dtheta_tmp[0].z;
    shmem[threadIdx.x].dthetaz[1] = dtheta_tmp[1].z;
    shmem[threadIdx.x].dthetaz[2] = dtheta_tmp[2].z;
    shmem[threadIdx.x].dthetaz[3] = dtheta_tmp[3].z;
    shmem[threadIdx.x].dthetaz[4] = dtheta_tmp[4].z;
    shmem[threadIdx.x].dthetaz[5] = dtheta_tmp[5].z;

  }
  __syncthreads();

  // We divide the 6x6x6 cube into 8 3x3x3 sub-cubes.
  // These sub-cubes are taken care by a single thread
  // 
  // Calculate the index this thread is calculating
  // tid = 0...63
  const int t = (tid % 8);         // 0...7
  // t = (tx0 + ty0*2 + tz0*4)/3
  // (tx0, ty0, tz0) gives the starting index of the 3x3x3 sub-cube
  const int tz0 = (t / 4)*3;
  const int ty0 = ((t / 2) % 2)*3;
  const int tx0 = (t % 2)*3;

  //
  // Calculate forces for 32 atoms. We have 32*2 = 64 threads
  // Loop is iterated 4 times:
  //                         (iterations)
  // Threads 0...7   = atoms 0, 8,  16, 24
  // Threads 8...15  = atoms 1, 9,  17, 25
  // Threads 16...31 = atoms 2, 10, 18, 26
  //                ...
  // Threads 56...63 = atoms 7, 15, 23, 31
  //

  int base = tid/8;
  const int base_end = pos_end - blockIdx.x*blockDim.x;
  while (base < base_end) {

    float f1 = 0.0f;
    float f2 = 0.0f;
    float f3 = 0.0f;
    int ix0 = shmem[base].ix;
    int iy0 = shmem[base].iy;
    int iz0 = shmem[base].iz;

    // Each thread calculates a 3x3x3 sub-cube
#pragma unroll
    for (int i=0;i < 27;i++) {
      int tz = tz0 + (i/9);
      int ty = ty0 + ((i/3) % 3);
      int tx = tx0 + (i % 3);

      int ix = ix0 + tx;
      int iy = iy0 + ty;
      int iz = iz0 + tz;
      if (ix >= nfftx) ix -= nfftx;
      if (iy >= nffty) iy -= nffty;
      if (iz >= nfftz) iz -= nfftz;
#ifdef USE_TEXTURE_OBJECTS
      float q0 = tex1Dfetch<float>(gridTexObj, ix + (iy + iz*ysize)*xsize);
#else
      float q0 = tex1Dfetch(gridTexRef, ix + (iy + iz*ysize)*xsize);
#endif
      float thx0 = shmem[base].thetax[tx];
      float thy0 = shmem[base].thetay[ty];
      float thz0 = shmem[base].thetaz[tz];
      float dthx0 = shmem[base].dthetax[tx];
      float dthy0 = shmem[base].dthetay[ty];
      float dthz0 = shmem[base].dthetaz[tz];
      f1 += dthx0 * thy0 * thz0 * q0;
      f2 += thx0 * dthy0 * thz0 * q0;
      f3 += thx0 * thy0 * dthz0 * q0;
    }

    //-------------------------

    // Reduce
#if __CUDA_ARCH__ >= 300
    const int i = threadIdx.x & 7;

    f1 += __shfl(f1, i+4, 8);
    f2 += __shfl(f2, i+4, 8);
    f3 += __shfl(f3, i+4, 8);

    f1 += __shfl(f1, i+2, 8);
    f2 += __shfl(f2, i+2, 8);
    f3 += __shfl(f3, i+2, 8);

    f1 += __shfl(f1, i+1, 8);
    f2 += __shfl(f2, i+1, 8);
    f3 += __shfl(f3, i+1, 8);

    if (i == 0) {
      shmem[base].f1 = f1;
      shmem[base].f2 = f2;
      shmem[base].f3 = f3;
    }

#else
    const int i = threadIdx.x & 7;
    shred[i].x = f1;
    shred[i].y = f2;
    shred[i].z = f3;

    if (i < 4) {
      shred[i].x += shred[i+4].x;
      shred[i].y += shred[i+4].y;
      shred[i].z += shred[i+4].z;
    }

    if (i < 2) {
      shred[i].x += shred[i+2].x;
      shred[i].y += shred[i+2].y;
      shred[i].z += shred[i+2].z;
    }

    if (i == 0) {
      shmem[base].f1 = shred[0].x + shred[1].x;
      shmem[base].f2 = shred[0].y + shred[1].y;
      shmem[base].f3 = shred[0].z + shred[1].z;
    }
#endif

    base += 8;
  }

  // Write forces
  __syncthreads();
  if (pos < pos_end && threadIdx.y == 0) {
    float f1 = shmem[threadIdx.x].f1;
    float f2 = shmem[threadIdx.x].f2;
    float f3 = shmem[threadIdx.x].f3;
    float q = shmem[threadIdx.x].charge*ccelec;
    float fx = q*recip1*f1*nfftx;
    float fy = q*recip2*f2*nffty;
    float fz = q*recip3*f3*nfftz;
    gather_force_store<FT>(fx, fy, fz, stride, pos, force);
  }

}

//
// Gathers forces from the grid
// blockDim.x            = Number of atoms each block loads
// blockDim.x*blockDim.y = Total number of threads per block
//
template <typename CT, typename FT>
__global__ void gather_force_8_ortho_kernel(const float4 *xyzq, const int ncoord,
					    const int nfftx, const int nffty, const int nfftz,
					    const int xsize, const int ysize, const int zsize,
					    const float recip1, const float recip2, const float recip3,
					    const float ccelec,
#ifdef USE_TEXTURE_OBJECTS
					    const hipTextureObject_t gridTexObj,
#endif
					    const int stride,
					    FT *force) {

  const int tid = threadIdx.x + threadIdx.y*blockDim.x; // 0...63

  // Shared memory
  __shared__ gather_t<CT, 8> shmem[32];
#if __CUDA_ARCH__ < 300
  __shared__ float3 shred_buf[32*2];
  volatile float3 *shred = &shred_buf[(tid/8)*8];
#endif

  const int pos = blockIdx.x*blockDim.x + threadIdx.x;
  const int pos_end = min((blockIdx.x+1)*blockDim.x, ncoord);

  // Load atom data into shared memory
  if (pos < pos_end && threadIdx.y == 0) {

    float4 xyzqi = xyzq[pos];
    float x = xyzqi.x;
    float y = xyzqi.y;
    float z = xyzqi.z;
    float q = xyzqi.w;

    float w;

    w = x*recip1 + 2.0f;
    float frx = (float)(nfftx*(w - (floorf(w + 0.5f) - 0.5f)));

    w = y*recip2 + 2.0f;
    float fry = (float)(nffty*(w - (floorf(w + 0.5f) - 0.5f)));

    w = z*recip3 + 2.0f;
    float frz = (float)(nfftz*(w - (floorf(w + 0.5f) - 0.5f)));

    int frxi = (int)frx;
    int fryi = (int)fry;
    int frzi = (int)frz;

    shmem[threadIdx.x].ix = frxi;
    shmem[threadIdx.x].iy = fryi;
    shmem[threadIdx.x].iz = frzi;
    shmem[threadIdx.x].charge = q;

    float wx = frx - (float)frxi;
    float wy = fry - (float)fryi;
    float wz = frz - (float)frzi;

    float3 theta_tmp[8];
    float3 dtheta_tmp[8];
    calc_theta_dtheta<float, float3, 8>(wx, wy, wz, theta_tmp, dtheta_tmp);

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].thetax[i] = theta_tmp[i].x;

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].thetay[i] = theta_tmp[i].y;

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].thetaz[i] = theta_tmp[i].z;

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].dthetax[i] = dtheta_tmp[i].x;

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].dthetay[i] = dtheta_tmp[i].y;

#pragma unroll
    for (int i=0;i < 8;i++)
      shmem[threadIdx.x].dthetaz[i] = dtheta_tmp[i].z;

  }
  __syncthreads();

  // We divide the 8x8x8 cube into 8 4x4x4 sub-cubes
  //
  // t = index this thread is calculating
  // tid = 0...63
  const int t = (tid % 8);         // 0...7
  // (tx0, ty0, tz0) = starting index of the 4x4x4 sub-cube
  const int tz0 = (t / 4)*4;
  const int ty0 = ((t / 2) % 2)*4;
  const int tx0 = (t % 2)*4;

  //
  // Calculate forces for 32 atoms. We have 32*2 = 64 threads
  // Loop is iterated 4 times:
  //                         (iterations)
  // Threads 0...7   = atoms 0, 8,  16, 24
  // Threads 8...15  = atoms 1, 9,  17, 25
  // Threads 16...31 = atoms 2, 10, 18, 26
  //                ...
  // Threads 56...63 = atoms 7, 15, 23, 31
  //

  int base = tid/8;
  const int base_end = pos_end - blockIdx.x*blockDim.x;
  while (base < base_end) {

    float f1 = 0.0f;
    float f2 = 0.0f;
    float f3 = 0.0f;
    int ix0 = shmem[base].ix;
    int iy0 = shmem[base].iy;
    int iz0 = shmem[base].iz;

    // Each thread calculates a 4x4x4 sub-cube
#pragma unroll
    for (int i=0;i < 64;i++) {
      int tz = tz0 + (i/16);
      int ty = ty0 + ((i/4) % 4);
      int tx = tx0 + (i % 4);

      int ix = ix0 + tx;
      int iy = iy0 + ty;
      int iz = iz0 + tz;
      if (ix >= nfftx) ix -= nfftx;
      if (iy >= nffty) iy -= nffty;
      if (iz >= nfftz) iz -= nfftz;
#ifdef USE_TEXTURE_OBJECTS
      float q0 = tex1Dfetch<float>(gridTexObj, ix + (iy + iz*ysize)*xsize);
#else
      float q0 = tex1Dfetch(gridTexRef, ix + (iy + iz*ysize)*xsize);
#endif
      float thx0 = shmem[base].thetax[tx];
      float thy0 = shmem[base].thetay[ty];
      float thz0 = shmem[base].thetaz[tz];
      float dthx0 = shmem[base].dthetax[tx];
      float dthy0 = shmem[base].dthetay[ty];
      float dthz0 = shmem[base].dthetaz[tz];
      f1 += dthx0 * thy0 * thz0 * q0;
      f2 += thx0 * dthy0 * thz0 * q0;
      f3 += thx0 * thy0 * dthz0 * q0;
    }

    //-------------------------

    // Reduce results from the 8 sub-cubes
#if __CUDA_ARCH__ >= 300
    const int i = threadIdx.x & 7;

    f1 += __shfl(f1, i+4, 8);
    f2 += __shfl(f2, i+4, 8);
    f3 += __shfl(f3, i+4, 8);

    f1 += __shfl(f1, i+2, 8);
    f2 += __shfl(f2, i+2, 8);
    f3 += __shfl(f3, i+2, 8);

    f1 += __shfl(f1, i+1, 8);
    f2 += __shfl(f2, i+1, 8);
    f3 += __shfl(f3, i+1, 8);

    if (i == 0) {
      shmem[base].f1 = f1;
      shmem[base].f2 = f2;
      shmem[base].f3 = f3;
    }

#else
    const int i = threadIdx.x & 7;
    shred[i].x = f1;
    shred[i].y = f2;
    shred[i].z = f3;

    if (i < 4) {
      shred[i].x += shred[i+4].x;
      shred[i].y += shred[i+4].y;
      shred[i].z += shred[i+4].z;
    }

    if (i < 2) {
      shred[i].x += shred[i+2].x;
      shred[i].y += shred[i+2].y;
      shred[i].z += shred[i+2].z;
    }

    if (i == 0) {
      shmem[base].f1 = shred[0].x + shred[1].x;
      shmem[base].f2 = shred[0].y + shred[1].y;
      shmem[base].f3 = shred[0].z + shred[1].z;
    }
#endif

    base += 8;
  }

  // Write forces
  __syncthreads();
  if (pos < pos_end && threadIdx.y == 0) {
    float f1 = shmem[threadIdx.x].f1;
    float f2 = shmem[threadIdx.x].f2;
    float f3 = shmem[threadIdx.x].f3;
    float q = shmem[threadIdx.x].charge*ccelec;
    float fx = q*recip1*f1*nfftx;
    float fy = q*recip2*f2*nffty;
    float fz = q*recip3*f3*nfftz;
    gather_force_store<FT>(fx, fy, fz, stride, pos, force);
  }

}

//
// Calculates self energy
// kappa_ccelec_sqrtpi = kappa*ccelec/sqrt(pi)
//
__global__ void calc_self_energy_kernel(const int ncoord, const float4* xyzq,
					const double kappa_ccelec_sqrtpi,
					double* __restrict__ energy_self) {
  // Shared memory
  // Required space: blockDim.x*sizeof(double)
  extern __shared__ double sh_q2[];

  int i = threadIdx.x + blockIdx.x*blockDim.x;
  float q = 0.0f;
  if (i < ncoord) q = xyzq[i].w;
  sh_q2[threadIdx.x] = q*q;
  __syncthreads();
  for(int d=1;d < blockDim.x;d *= 2) {
    int t = threadIdx.x + d;
    double q2_val = (t < blockDim.x) ? sh_q2[t] : 0.0;
    __syncthreads();
    sh_q2[threadIdx.x] += q2_val;
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    atomicAdd(energy_self, -sh_q2[0]*kappa_ccelec_sqrtpi);
  }

}

//#####################################################################################
//#####################################################################################
//#####################################################################################

template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::setup_grid_texture(CT *data, const int data_len) {
  if (sizeof(CT) != 4) {
    std::cerr << "CudaPMERecip::setup_grid_texture, current implementation only tested for float-type textures" << std::endl;
    exit(1);
  }
#ifdef USE_TEXTURE_OBJECTS
  // Use texture objects
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = data;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = sizeof(CT)*8;
  resDesc.res.linear.sizeInBytes = data_len*sizeof(CT);
  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;
  cudaCheck(hipCreateTextureObject(&gridTexObj, &resDesc, &texDesc, NULL));
#else
  gridTexRef.normalized = 0;
  gridTexRef.filterMode = hipFilterModePoint;
  gridTexRef.addressMode[0] = hipAddressModeClamp;
  gridTexRef.channelDesc.x = sizeof(CT)*8;
  gridTexRef.channelDesc.y = 0;
  gridTexRef.channelDesc.z = 0;
  gridTexRef.channelDesc.w = 0;
  gridTexRef.channelDesc.f = hipChannelFormatKindFloat;
  cudaCheck(hipBindTexture(NULL, gridTexRef, data, data_len*sizeof(CT)));
#endif
}

//
// Initializer
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::init(int x0, int x1, int y0, int y1, int z0, int z1, int order, 
				     bool y_land_locked, bool z_land_locked) {
  
  this->x0 = x0;
  this->x1 = x1;
  
  this->y0 = y0;
  this->y1 = y1;
  
  this->z0 = z0;
  this->z1 = z1;
  
  this->order = order;
  
  xlo = x0;
  xhi = x1;

  ylo = y0;
  yhi = y1;

  zlo = z0;
  zhi = z1;

  /*
  xhi += (order-1);

  if (y_land_locked) ylo -= (order-1);
  yhi += (order-1);
  
  if (z_land_locked) zlo -= (order-1);
  zhi += (order-1);
  */

  xsize = xhi - xlo + 1;
  ysize = yhi - ylo + 1;
  zsize = zhi - zlo + 1;

  data_size = (2*(xsize/2+1))*ysize*zsize;

  make_fft_plans();
  set_stream(stream);

  // data1 is used for accumulation, make sure it has enough space
  allocate<CT>(&data1, data_size*sizeof(AT)/sizeof(CT));
  allocate<CT>(&data2, data_size);

  if (multi_gpu) {
#if CUDA_VERSION >= 6000
    cufftCheck(hipfftXtMalloc(r2c_plan, &multi_data, HIPFFT_XT_FORMAT_INPLACE));
    host_data = new CT2[xsize*ysize*zsize];
    host_tmp = new CT[2*(xsize/2+1)*ysize*zsize];
#else
    std::cerr << "No Multi-gpu FFT support in CUDA versions below 6.0" << std::endl;
    exit(1);
#endif
  }

  data1_len = data_size*sizeof(AT)/sizeof(CT);
  data2_len = data_size;

  accum_grid  = new Matrix3d<AT>(xsize, ysize, zsize, xsize, ysize, zsize, (AT *)data1);
  charge_grid = new Matrix3d<CT>(xsize, ysize, zsize, xsize, ysize, zsize, (CT *)data2);

  if (fft_type == COLUMN) {
    xfft_grid   = new Matrix3d<CT2>(xsize/2+1, ysize, zsize, xsize/2+1, ysize, zsize, (CT2 *)data2);
    yfft_grid   = new Matrix3d<CT2>(ysize, zsize, xsize/2+1, ysize, zsize, xsize/2+1, (CT2 *)data1);
    zfft_grid   = new Matrix3d<CT2>(zsize, xsize/2+1, ysize, zsize, xsize/2+1, ysize, (CT2 *)data2);
    solved_grid = new Matrix3d<CT>(xsize, ysize, zsize, xsize, ysize, zsize, (CT *)data2);
  } else if (fft_type == SLAB) {
    xyfft_grid = new Matrix3d<CT2>(xsize/2+1, ysize, zsize, xsize/2+1, ysize, zsize, (CT2 *)data2);
    zfft_grid   = new Matrix3d<CT2>(zsize, xsize/2+1, ysize, zsize, xsize/2+1, ysize, (CT2 *)data1);
    solved_grid = new Matrix3d<CT>(xsize, ysize, zsize, xsize, ysize, zsize, (CT *)data2);
  } else if (fft_type == BOX) {
    fft_grid = new Matrix3d<CT2>(xsize/2+1, ysize, zsize, xsize/2+1, ysize, zsize, (CT2 *)data2);
    solved_grid = new Matrix3d<CT>(xsize, ysize, zsize, xsize, ysize, zsize, (CT *)data2);
  }

  // Bind grid_texture to solved_grid->data (data2)
  setup_grid_texture(solved_grid->data, xsize*ysize*zsize);

}

//
// Class creator 
//
template <typename AT, typename CT, typename CT2>
CudaPMERecip<AT, CT, CT2>::CudaPMERecip(int nfftx, int nffty, int nfftz, int order,
					FFTtype fft_type, int nnode, int mynode,
					CudaEnergyVirial& energyVirial, const char* nameRecip, const char* nameSelf,
					hipStream_t stream) :
  nfftx(nfftx), nffty(nffty), nfftz(nfftz), fft_type(fft_type),
  energyVirial(energyVirial), stream(stream) {

  assert(nnode >= 1);
  assert(mynode >= 0 && mynode < nnode);
  assert(sizeof(AT) >= sizeof(CT));
  assert(nameRecip != NULL);
  assert(nameSelf != NULL);

  // Insert energy terms
  energyVirial.insert(nameRecip);
  strRecip = nameRecip;
  energyVirial.insert(nameSelf);
  strSelf = nameSelf;
  
  int nnode_y, nnode_z;
  
  if (fft_type == COLUMN) {
    nnode_y = max(1,(int)ceil( sqrt( (double)(nnode*nffty) / (double)(nfftz) )));
    nnode_z = nnode/nnode_y;
    while (nnode_y*nnode_z != nnode) {
      nnode_y = nnode_y - 1;
      nnode_z = nnode/nnode_y;
    }
  } else if (fft_type == SLAB) {
    nnode_y = 1;
    nnode_z = nnode;
    assert(nfftz/nnode_z >= 1);
  } else if (fft_type == BOX) {
    assert(nnode == 1);
    nnode_y = 1;
    nnode_z = 1;
  } else {
    std::cerr<<"CudaPMERecip::fft_type invalid"<<std::endl;
    exit(1);
  }

  // We have nodes nnode_y * nnode_z. Get y and z index of this node:
  int inode_y = mynode % nnode_y;
  int inode_z = mynode/nnode_y;

  assert(nnode_y != 0);
  assert(nnode_z != 0);

  int x0 = 0;
  int x1 = nfftx-1;
      
  int y0 = inode_y*nffty/nnode_y;
  int y1 = (inode_y+1)*nffty/nnode_y - 1;

  int z0 = inode_z*nfftz/nnode_z;
  int z1 = (inode_z+1)*nfftz/nnode_z - 1;

  bool y_land_locked = (inode_y-1 >= 0) && (inode_y+1 < nnode_y);
  bool z_land_locked = (inode_z-1 >= 0) && (inode_z+1 < nnode_z);

  multi_gpu = false;

  assert((multi_gpu && fft_type==BOX) || !multi_gpu);

  init(x0, x1, y0, y1, z0, z1, order, y_land_locked, z_land_locked);

  allocate<CT>(&prefac_x, nfftx);
  allocate<CT>(&prefac_y, nffty);
  allocate<CT>(&prefac_z, nfftz);
  calc_prefac();

  //allocate<RecipEnergyVirial_t>(&d_energy_virial, 1);
  //allocate_host<RecipEnergyVirial_t>(&h_energy_virial, 1);

  //clear_energy_virial();
}

//
// Create FFT plans
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::make_fft_plans() {

  if (fft_type == COLUMN) {
    // Set the size of the local FFT transforms
    int batch;
    int nfftx_local = x1 - x0 + 1;
    int nffty_local = y1 - y0 + 1;
    int nfftz_local = z1 - z0 + 1;
    
    batch = nffty_local * nfftz_local;
    cufftCheck(hipfftPlanMany(&x_r2c_plan, 1, &nfftx_local,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_R2C, batch));
    cufftCheck(cufftSetCompatibilityMode(x_r2c_plan, CUFFT_COMPATIBILITY_NATIVE));
    
    batch = nfftz_local*(nfftx_local/2+1);
    cufftCheck(hipfftPlanMany(&y_c2c_plan, 1, &nffty_local,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_C2C, batch));
    cufftCheck(cufftSetCompatibilityMode(y_c2c_plan, CUFFT_COMPATIBILITY_NATIVE));

    batch = (nfftx_local/2+1)*nffty_local;
    cufftCheck(hipfftPlanMany(&z_c2c_plan, 1, &nfftz_local,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_C2C, batch));
    cufftCheck(cufftSetCompatibilityMode(z_c2c_plan, CUFFT_COMPATIBILITY_NATIVE));

    batch = nffty_local*nfftz_local;
    cufftCheck(hipfftPlanMany(&x_c2r_plan, 1, &nfftx_local,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_C2R, batch));
    cufftCheck(cufftSetCompatibilityMode(x_c2r_plan, CUFFT_COMPATIBILITY_NATIVE));
  } else if (fft_type == SLAB) {
    int batch;
    int nfftx_local = x1 - x0 + 1;
    int nffty_local = y1 - y0 + 1;
    int nfftz_local = z1 - z0 + 1;

    int n[2] = {nffty_local, nfftx_local};

    batch = nfftz_local;
    cufftCheck(hipfftPlanMany(&xy_r2c_plan, 2, n,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_R2C, batch));
    cufftCheck(cufftSetCompatibilityMode(xy_r2c_plan, CUFFT_COMPATIBILITY_NATIVE));

    batch = (nfftx_local/2+1)*nffty_local;
    cufftCheck(hipfftPlanMany(&z_c2c_plan, 1, &nfftz_local,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_C2C, batch));
    cufftCheck(cufftSetCompatibilityMode(z_c2c_plan, CUFFT_COMPATIBILITY_NATIVE));

    batch = nfftz_local;
    cufftCheck(hipfftPlanMany(&xy_c2r_plan, 2, n,
			     NULL, 0, 0,
			     NULL, 0, 0, 
			     HIPFFT_C2R, batch));
    cufftCheck(cufftSetCompatibilityMode(xy_c2r_plan, CUFFT_COMPATIBILITY_NATIVE));
    
  } else if (fft_type == BOX) {
    if (multi_gpu) {
#if CUDA_VERSION >= 6000
      cufftCheck(hipfftCreate(&r2c_plan));
      cufftCheck(hipfftCreate(&c2r_plan));
      int ngpu = 2;
      int gpu[2] = {2, 3};
      cufftCheck(hipfftXtSetGPUs(r2c_plan, ngpu, gpu));
      cufftCheck(hipfftXtSetGPUs(c2r_plan, ngpu, gpu));

      size_t worksize_r2c[2];
      size_t worksize_c2r[2];

      cufftCheck(hipfftMakePlan3d(r2c_plan, nfftz, nffty, nfftx, HIPFFT_C2C, worksize_r2c));
      cufftCheck(hipfftMakePlan3d(c2r_plan, nfftz, nffty, nfftx, HIPFFT_C2C, worksize_c2r));
#endif
    } else {
      cufftCheck(hipfftPlan3d(&r2c_plan, nfftz, nffty, nfftx, HIPFFT_R2C));
      cufftCheck(cufftSetCompatibilityMode(r2c_plan, CUFFT_COMPATIBILITY_NATIVE));

      cufftCheck(hipfftPlan3d(&c2r_plan, nfftz, nffty, nfftx, HIPFFT_C2R));
      cufftCheck(cufftSetCompatibilityMode(c2r_plan, CUFFT_COMPATIBILITY_NATIVE));
    }
  }

}

//
// Set stream
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::set_stream(hipStream_t stream) {

  this->stream = stream;

  if (fft_type == COLUMN) {
    cufftCheck(hipfftSetStream(x_r2c_plan, stream));
    cufftCheck(hipfftSetStream(y_c2c_plan, stream));
    cufftCheck(hipfftSetStream(z_c2c_plan, stream));    
    cufftCheck(hipfftSetStream(x_c2r_plan, stream));
  } else if (fft_type == SLAB) {
    cufftCheck(hipfftSetStream(xy_r2c_plan, stream));
    cufftCheck(hipfftSetStream(z_c2c_plan, stream));
    cufftCheck(hipfftSetStream(xy_c2r_plan, stream));
  } else if (fft_type == BOX) {
    cufftCheck(hipfftSetStream(r2c_plan, stream));
    cufftCheck(hipfftSetStream(c2r_plan, stream));
  }

}


//
// Class destructor
//
template <typename AT, typename CT, typename CT2>
CudaPMERecip<AT, CT, CT2>::~CudaPMERecip() {

#ifdef USE_TEXTURE_OBJECTS
  cudaCheck(hipDestroyTextureObject(gridTexObj));
#else
  // Unbind grid texture
  cudaCheck(hipUnbindTexture(gridTexRef));
#endif
  
  delete accum_grid;
  delete charge_grid;
  delete solved_grid;
  deallocate<CT>(&data1);
  deallocate<CT>(&data2);

#if CUDA_VERSION >= 6000
  if (multi_gpu) {
    delete [] host_data;
    delete [] host_tmp;
    cufftCheck(hipfftXtFree(multi_data));
  }
#endif

  if (fft_type == COLUMN) {
    delete xfft_grid;
    delete yfft_grid;
    delete zfft_grid;
    cufftCheck(hipfftDestroy(x_r2c_plan));
    cufftCheck(hipfftDestroy(y_c2c_plan));
    cufftCheck(hipfftDestroy(z_c2c_plan));
    cufftCheck(hipfftDestroy(x_c2r_plan));
  } else if (fft_type == SLAB) {
    delete xyfft_grid;
    delete zfft_grid;
    cufftCheck(hipfftDestroy(xy_r2c_plan));
    cufftCheck(hipfftDestroy(z_c2c_plan));
    cufftCheck(hipfftDestroy(xy_c2r_plan));
  } else if (fft_type == BOX) {
    delete fft_grid;
    cufftCheck(hipfftDestroy(r2c_plan));
    cufftCheck(hipfftDestroy(c2r_plan));
  }

  deallocate<CT>(&prefac_x);
  deallocate<CT>(&prefac_y);
  deallocate<CT>(&prefac_z);

  //if (d_energy_virial != NULL) deallocate<RecipEnergyVirial_t>(&d_energy_virial);
  //if (h_energy_virial != NULL) deallocate_host<RecipEnergyVirial_t>(&h_energy_virial);
}

template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::print_info() {
  std::cout << "fft_type = ";
  if (fft_type == COLUMN) {
    std::cout << "COLUMN" << std::endl;
  } else if (fft_type == SLAB) {
    std::cout << "SLAB" << std::endl;
  } else {
    std::cout << "BOX" << std::endl;
  }
  std::cout << "order = " << order << std::endl;
  std::cout << "nfftx, nffty, nfftz = " << nfftx << " " << nffty << " " << nfftz << std::endl;
  std::cout << "x0...x1   = " << x0 << " ... " << x1 << std::endl;
  std::cout << "y0...y1   = " << y0 << " ... " << y1 << std::endl;
  std::cout << "z0...z1   = " << z0 << " ... " << z1 << std::endl;
  std::cout << "xlo...xhi = " << xlo << " ... " << xhi << std::endl;
  std::cout << "ylo...yhi = " << ylo << " ... " << yhi << std::endl;
  std::cout << "zlo...zhi = " << zlo << " ... " << zhi << std::endl;
  std::cout << "xsize, ysize, zsize = " << xsize << " " << ysize << " " << zsize << std::endl;
  std::cout << "data_size = " << data_size << std::endl;
}

//
// Spreads charge on grid. Uses pre-calculated B-splines (slower)
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::spread_charge(const int ncoord, const Bspline<CT> &bspline) {

  clear_gpu_array<AT>((AT *)accum_grid->data, xsize*ysize*zsize, stream);

  dim3 nthread, nblock;

  nthread.x = 32;
  nthread.y = 4;
  nthread.z = 1;
  nblock.x = (ncoord - 1)/nthread.x + 1;
  nblock.y = 1;
  nblock.z = 1;

  size_t shmem_size = sizeof(spread_t)*nthread.x;

  switch(order) {
  case 4:
    spread_charge_4<AT> <<< nblock, nthread, shmem_size, stream >>>
      (ncoord, 
       bspline.gix, bspline.giy, bspline.giz,
       bspline.charge,
       (float4 *)bspline.thetax,
       (float4 *)bspline.thetay,
       (float4 *)bspline.thetaz,
       nfftx, nffty, nfftz,
       (AT *)accum_grid->data);
    break;

  default:
    std::cerr<<"CudaPMERecip::spread_charge: order "<<order<<" not implemented"<<std::endl;
    exit(1);
  }
  cudaCheck(hipGetLastError());

  // Reduce charge data back to a float/double value
  nthread.x = 512;
  nthread.y = 1;
  nthread.z = 1;
  nblock.x = (nfftx*nffty*nfftz - 1)/nthread.x + 1;
  nblock.y = 1;
  nblock.z = 1;
  reduce_force<AT, CT> <<< nblock, nthread, 0, stream >>>(xsize*ysize*zsize,
							  (AT *)accum_grid->data,
							  charge_grid->data);
  cudaCheck(hipGetLastError());

}

//
// Spreads charge on grid. Calculates B-splines on-the-fly (faster)
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::spread_charge(const float4 *xyzq, const int ncoord, const double *recip) {

  clear_gpu_array<AT>((AT *)accum_grid->data, xsize*ysize*zsize, stream);

  dim3 nthread, nblock;

  CT recip1 = (CT)recip[0];
  CT recip2 = (CT)recip[4];
  CT recip3 = (CT)recip[8];

  switch(order) {
  case 4:
    nthread.x = 32;
    nthread.y = 4;
    nthread.z = 1;
    nblock.x = (ncoord - 1)/nthread.x + 1;
    nblock.y = 1;
    nblock.z = 1;
    spread_charge_ortho_4<AT> <<< nblock, nthread, 0, stream >>>
      (xyzq, ncoord, recip1, recip2, recip3,
       nfftx, nffty, nfftz,
       (AT *)accum_grid->data);
    break;

  case 6:
    nthread.x = 32;
    nthread.y = 7;
    nthread.z = 1;
    nblock.x = (ncoord - 1)/nthread.x + 1;
    nblock.y = 1;
    nblock.z = 1;
    spread_charge_ortho_6<AT> <<< nblock, nthread, 0, stream >>>
      (xyzq, ncoord, recip1, recip2, recip3,
       nfftx, nffty, nfftz,
       (AT *)accum_grid->data);
    break;

  case 8:
    nthread.x = 32;
    nthread.y = 16;
    nthread.z = 1;
    nblock.x = (ncoord - 1)/nthread.x + 1;
    nblock.y = 1;
    nblock.z = 1;
    spread_charge_ortho_8<AT> <<< nblock, nthread, 0, stream >>>
      (xyzq, ncoord, recip1, recip2, recip3,
       nfftx, nffty, nfftz,
       (AT *)accum_grid->data);
    break;

  default:
    std::cerr<<"CudaPMERecip::spread_charge: order "<<order<<" not implemented"<<std::endl;
    exit(1);
  }
  cudaCheck(hipGetLastError());

  // Reduce charge data back to a float/double value
  nthread.x = 512;
  nthread.y = 1;
  nthread.z = 1;
  nblock.x = (nfftx*nffty*nfftz - 1)/nthread.x + 1;
  nblock.y = 1;
  nblock.z = 1;
  reduce_force<AT, CT> <<< nblock, nthread, 0, stream >>>
    (xsize*ysize*zsize,
     (AT *)accum_grid->data,
     charge_grid->data);

  cudaCheck(hipGetLastError());

}

//
// Perform scalar sum
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::scalar_sum(const double *recip, const double kappa,
					   const bool calc_energy, const bool calc_virial) {

  bool calc_energy_virial = (calc_energy || calc_virial);

  // Best performance:
  // cuda_arch = 200:
  // energy & virial & (C2075 | K40c) & 512x14: 102.7 (C2075) | 70.4 (K240c)
  // C2075 & 768x12: 27.4

  int nthread = 512;
  int nblock = 10;

  if (get_cuda_arch() < 300) {
    if (calc_energy_virial) {
      nthread = 512;
      nblock = 14;
    } else {
      nthread = 768;
      nblock = 12;
    }
  } else {
    if (calc_energy_virial) {
      nthread = 1024;
      nblock = 14;
    } else {
      nthread = 1024;
      nblock = 14;
    }
  }

  int shmem_size = sizeof(CT)*(nfftx + nffty + nfftz);
  if (calc_energy_virial) {
    if (get_cuda_arch() < 300) {
      shmem_size = max(shmem_size, (int)(nthread*sizeof(RecipVirial_t)));
    } else {
      shmem_size = max(shmem_size, (int)((nthread/warpsize)*sizeof(RecipVirial_t)));
    }
  }

  int nfft1, nfft2, nfft3;
  int size1, size2, size3;
  CT *prefac1, *prefac2, *prefac3;
  CT recip1, recip2, recip3;
  CT2 *datap;

  if (fft_type == COLUMN || fft_type == SLAB) {
    nfft1 = nfftz;
    nfft2 = nfftx;
    nfft3 = nffty;
    size1 = nfftz;
    size2 = nfftx/2+1;
    size3 = nffty;
    prefac1 = prefac_z;
    prefac2 = prefac_x;
    prefac3 = prefac_y;
    recip1 = (CT)recip[8];
    recip2 = (CT)recip[0];
    recip3 = (CT)recip[4];
    datap = zfft_grid->data;
  } else if (fft_type == BOX) {
    nfft1 = nfftx;
    nfft2 = nffty;
    nfft3 = nfftz;
    size1 = nfftx/2+1;
    size2 = nffty;
    size3 = nfftz;
    prefac1 = prefac_x;
    prefac2 = prefac_y;
    prefac3 = prefac_z;
    recip1 = (CT)recip[0];
    recip2 = (CT)recip[4];
    recip3 = (CT)recip[8];
    datap = fft_grid->data;
  }

  bool ortho = (recip[1] == 0.0 && recip[2] == 0.0 && recip[3] == 0.0 &&
		recip[5] == 0.0 && recip[6] == 0.0 && recip[7] == 0.0);

  double inv_vol = recip[0]*recip[4]*recip[8];
  CT piv_inv = (CT)(inv_vol/pi);
  CT fac = (CT)(pi*pi/(kappa*kappa));

  bool global_base = (x0 == 0 && y0 == 0 && z0 == 0);

  int nf1 = nfft1/2 + (nfft1 % 2);
  int nf2 = nfft2/2 + (nfft2 % 2);
  int nf3 = nfft3/2 + (nfft3 % 2);

  if (ortho) {
    if (calc_energy_virial) {
      scalar_sum_ortho_kernel<CT, CT2, true>
	<<< nblock, nthread, shmem_size, stream >>>
	(nfft1, nfft2, nfft3,
	 size1, size2, size3,
	 nf1, nf2, nf3,
	 recip1, recip2, recip3,
	 prefac1, prefac2, prefac3,
	 fac, piv_inv, global_base, datap,
	 energyVirial.getEnergyPointer(strRecip),
	 energyVirial.getVirialPointer());
      cudaCheck(hipGetLastError());
    } else {
      scalar_sum_ortho_kernel<CT, CT2, false>
	<<< nblock, nthread, shmem_size, stream >>>
	(nfft1, nfft2, nfft3,
	 size1, size2, size3,
	 nf1, nf2, nf3,
	 recip1, recip2, recip3,
	 prefac1, prefac2, prefac3,
	 fac, piv_inv, global_base, datap, NULL, NULL);
      cudaCheck(hipGetLastError());
    }
  } else {
    std::cerr<<"CudaPMERecip::scalar_sum: only orthorombic boxes are currently supported"<<std::endl;
    exit(1);
  }

}

//
// Calculates self energy
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::calc_self_energy(const float4 *xyzq, const int ncoord, const double kappa) {
  int nthread = 256;
  int nblock = (ncoord-1)/nthread+1;
  int shmem_size = nthread*sizeof(double);
  double kappa_ccelec_sqrtpi = kappa*ccelec/sqrt(pi);
  calc_self_energy_kernel<<< nblock, nthread, shmem_size, stream >>>
    (ncoord, xyzq, kappa_ccelec_sqrtpi, energyVirial.getEnergyPointer(strSelf));
  cudaCheck(hipGetLastError());
}

//
// Gathers forces from the grid
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::gather_force(const int ncoord, const double* recip,
				     const Bspline<CT> &bspline,
				     const int stride, CT* force) {

  dim3 nthread(32, 2, 1);
  dim3 nblock((ncoord - 1)/nthread.x + 1, 1, 1);
  size_t shmem_size = sizeof(gather_t<CT, 4>)*nthread.x + sizeof(float3)*nthread.x*nthread.y;

  CT recip_loc[9];
  recip_loc[0] = (CT)(recip[0]*(double)nfftx*ccelec);
  recip_loc[1] = (CT)(recip[1]*(double)nfftx*ccelec);
  recip_loc[2] = (CT)(recip[2]*(double)nfftx*ccelec);
  recip_loc[3] = (CT)(recip[3]*(double)nffty*ccelec);
  recip_loc[4] = (CT)(recip[4]*(double)nffty*ccelec);
  recip_loc[5] = (CT)(recip[5]*(double)nffty*ccelec);
  recip_loc[6] = (CT)(recip[6]*(double)nfftz*ccelec);
  recip_loc[7] = (CT)(recip[7]*(double)nfftz*ccelec);
  recip_loc[8] = (CT)(recip[8]*(double)nfftz*ccelec);

  bool ortho = (recip[1] == 0.0 && recip[2] == 0.0 && recip[3] == 0.0 &&
		recip[5] == 0.0 && recip[6] == 0.0 && recip[7] == 0.0);

  if (ortho) {
    switch(order) {
    case 4:
      gather_force_4_ortho_kernel<CT> 
	<<< nblock, nthread, shmem_size, stream >>>
	(ncoord,
	 nfftx, nffty, nfftz,
	 nfftx, nffty, nfftz,
	 recip_loc[0], recip_loc[4], recip_loc[8],
	 bspline.gix, bspline.giy, bspline.giz, bspline.charge,
	 (float4 *)bspline.thetax,
	 (float4 *)bspline.thetay,
	 (float4 *)bspline.thetaz,
	 (float4 *)bspline.dthetax,
	 (float4 *)bspline.dthetay,
	 (float4 *)bspline.dthetaz,
#ifdef USE_TEXTURE_OBJECTS
	 gridTexObj,
#endif
	 stride, force);
      break;

    default:
      std::cerr<<"CudaPMERecip::gather_force: order "<<order<<" not implemented"<<std::endl;
      exit(1);
    }
  } else {
      std::cerr<<"CudaPMERecip::gather_force: only orthorombic boxes are currently supported"<<std::endl;
      std::cerr<<recip[1]<<std::endl;
      std::cerr<<recip[2]<<std::endl;
      std::cerr<<recip[3]<<std::endl;
      std::cerr<<recip[5]<<std::endl;
      std::cerr<<recip[6]<<std::endl;
      std::cerr<<recip[7]<<std::endl;
      exit(1);    
  }

  cudaCheck(hipGetLastError());
}

//
// Gathers forces from the grid
//
template <typename AT, typename CT, typename CT2>
template <typename FT>
void CudaPMERecip<AT, CT, CT2>::gather_force(const float4 *xyzq, const int ncoord, const double* recip,
				     const int stride, FT* force) {

  dim3 nthread(32, 2, 1);
  dim3 nblock((ncoord - 1)/nthread.x + 1, 1, 1);
  //size_t shmem_size = sizeof(gather_t<CT>)*nthread.x;// + sizeof(float3)*nthread.x*nthread.y;

  CT recip_loc[9];
  recip_loc[0] = (CT)(recip[0]);
  recip_loc[1] = (CT)(recip[1]);
  recip_loc[2] = (CT)(recip[2]);
  recip_loc[3] = (CT)(recip[3]);
  recip_loc[4] = (CT)(recip[4]);
  recip_loc[5] = (CT)(recip[5]);
  recip_loc[6] = (CT)(recip[6]);
  recip_loc[7] = (CT)(recip[7]);
  recip_loc[8] = (CT)(recip[8]);

  CT ccelec_loc = (CT)ccelec;

  bool ortho = (recip[1] == 0.0 && recip[2] == 0.0 && recip[3] == 0.0 &&
		recip[5] == 0.0 && recip[6] == 0.0 && recip[7] == 0.0);

  if (ortho) {
    switch(order) {
    case 4:
      gather_force_4_ortho_kernel<CT, FT> 
	<<< nblock, nthread, 0, stream >>>
	(xyzq, ncoord,
	 nfftx, nffty, nfftz,
	 nfftx, nffty, nfftz,
	 recip_loc[0], recip_loc[4], recip_loc[8],
	 ccelec_loc,
#ifdef USE_TEXTURE_OBJECTS
	 gridTexObj,
#endif
	 stride, force);
      break;

    case 6:
      gather_force_6_ortho_kernel<CT, FT> 
	<<< nblock, nthread, 0, stream >>>
	(xyzq, ncoord,
	 nfftx, nffty, nfftz,
	 nfftx, nffty, nfftz,
	 recip_loc[0], recip_loc[4], recip_loc[8],
	 ccelec_loc,
#ifdef USE_TEXTURE_OBJECTS
	 gridTexObj,
#endif
	 stride, force);
      break;
 
    case 8:
      gather_force_8_ortho_kernel<CT, FT> 
	<<< nblock, nthread, 0, stream >>>
	(xyzq, ncoord,
	 nfftx, nffty, nfftz,
	 nfftx, nffty, nfftz,
	 recip_loc[0], recip_loc[4], recip_loc[8],
	 ccelec_loc,
#ifdef USE_TEXTURE_OBJECTS
	 gridTexObj,
#endif
	 stride, force);
      break;

    default:
      std::cerr<<"CudaPMERecip::gather_force: order "<<order<<" not implemented"<<std::endl;
      exit(1);
    }
  } else {
      std::cerr<<"CudaPMERecip::gather_force: only orthorombic boxes are currently supported"<<std::endl;
      std::cerr<<recip[1]<<std::endl;
      std::cerr<<recip[2]<<std::endl;
      std::cerr<<recip[3]<<std::endl;
      std::cerr<<recip[5]<<std::endl;
      std::cerr<<recip[6]<<std::endl;
      std::cerr<<recip[7]<<std::endl;
      exit(1);    
  }

  cudaCheck(hipGetLastError());
}

//
// FFT x coordinate Real -> Complex
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::x_fft_r2c(CT2 *data) {

  if (fft_type == COLUMN) {
    cufftCheck(hipfftExecR2C(x_r2c_plan,
			    (hipfftReal *)data,
			    (hipfftComplex *)data));
  } else {
    std::cerr << "CudaPMERecip::x_fft_r2c, only COLUMN type FFT can call this function" << std::endl;
    exit(1);
  }

}

//
// FFT x coordinate Complex -> Real
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::x_fft_c2r(CT2 *data) {

  if (fft_type == COLUMN) {
    cufftCheck(hipfftExecC2R(x_c2r_plan,
			    (hipfftComplex *)data,
			    (hipfftReal *)data));
  } else {
    std::cerr << "CudaPMERecip::x_fft_r2c, only COLUMN type FFT can call this function" << std::endl;
    exit(1);
  }

}

//
// FFT y coordinate Complex -> Complex
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::y_fft_c2c(CT2 *data, const int direction) {

  if (fft_type == COLUMN) {
    cufftCheck(hipfftExecC2C(y_c2c_plan,
			    (hipfftComplex *)data,
			    (hipfftComplex *)data,
			    direction));
  } else {
    std::cerr << "CudaPMERecip::x_fft_r2c, only COLUMN type FFT can call this function" << std::endl;
    exit(1);
  }

}

//
// FFT z coordinate Complex -> Complex
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::z_fft_c2c(CT2 *data, const int direction) {

  if (fft_type == COLUMN) {
    cufftCheck(hipfftExecC2C(z_c2c_plan,
			    (hipfftComplex *)data,
			    (hipfftComplex *)data,
			    direction));
  } else {
    std::cerr << "CudaPMERecip::x_fft_r2c, only COLUMN type FFT can call this function" << std::endl;
    exit(1);
  }

}

//
// 3D FFT Real -> Complex
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::r2c_fft() {

  if (fft_type == COLUMN) {
    // data2(x, y, z)
    x_fft_r2c(xfft_grid->data);
    xfft_grid->transpose_xyz_yzx(yfft_grid);

    // data1(y, z, x)
    y_fft_c2c(yfft_grid->data, HIPFFT_FORWARD);
    yfft_grid->transpose_xyz_yzx(zfft_grid);

    // data2(z, x, y)
    z_fft_c2c(zfft_grid->data, HIPFFT_FORWARD);
  } else if (fft_type == SLAB) {
    cufftCheck(hipfftExecR2C(xy_r2c_plan,
			    (hipfftReal *)charge_grid->data,
			    (hipfftComplex *)xyfft_grid->data));
    xyfft_grid->transpose_xyz_zxy(zfft_grid);
    cufftCheck(hipfftExecC2C(z_c2c_plan,
			    (hipfftComplex *)zfft_grid->data,
			    (hipfftComplex *)zfft_grid->data, HIPFFT_FORWARD));
  } else if (fft_type == BOX) {
    if (multi_gpu) {
#if CUDA_VERSION >= 6000
      // Transform from Real -> Complex
      cudaCheck(hipMemcpy(host_tmp, charge_grid->data, sizeof(CT)*xsize*ysize*zsize,
			   hipMemcpyDeviceToHost));
      for (int z=0;z < zsize;z++)
	for (int y=0;y < ysize;y++)
	  for (int x=0;x < xsize;x++) {
	    host_data[x + (y + z*ysize)*xsize].x = host_tmp[x + (y + z*ysize)*xsize];
	    host_data[x + (y + z*ysize)*xsize].y = 0;
	  }


      cufftCheck(hipfftXtMemcpy(r2c_plan, multi_data, host_data, HIPFFT_COPY_HOST_TO_DEVICE));
      cufftCheck(hipfftXtExecDescriptorC2C(r2c_plan,
					  multi_data,
					  multi_data, HIPFFT_FORWARD));
      // Copy data back to a single GPU buffer in fft_grid->data
      cufftCheck(hipfftXtMemcpy(r2c_plan, host_data, multi_data, HIPFFT_COPY_DEVICE_TO_HOST));

      CT2 *tmp = (CT2 *)host_tmp;
      for (int z=0;z < zsize;z++)
	for (int y=0;y < ysize;y++)
	  for (int x=0;x < xsize/2+1;x++) {
	    tmp[x + (y + z*ysize)*(xsize/2+1)].x = host_data[x + (y + z*ysize)*xsize].x;
	    tmp[x + (y + z*ysize)*(xsize/2+1)].y = host_data[x + (y + z*ysize)*xsize].y;
	  }

      cudaCheck(hipMemcpy(fft_grid->data, tmp, sizeof(CT2)*(xsize/2+1)*ysize*zsize,
			   hipMemcpyHostToDevice));
#endif
    } else {
      cufftCheck(hipfftExecR2C(r2c_plan,
			      (hipfftReal *)charge_grid->data,
			      (hipfftComplex *)fft_grid->data));
    }
  }

}

//
// 3D FFT Complex -> Real
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::c2r_fft() {

  if (fft_type == COLUMN) {
    // data2(z, x, y)
    z_fft_c2c(zfft_grid->data, HIPFFT_BACKWARD);
    zfft_grid->transpose_xyz_zxy(yfft_grid);

    // data1(y, x, z)
    y_fft_c2c(yfft_grid->data, HIPFFT_BACKWARD);
    yfft_grid->transpose_xyz_zxy(xfft_grid);

    // data2(x, y, z)
    x_fft_c2r(xfft_grid->data);
  } else if (fft_type == SLAB) {
    cufftCheck(hipfftExecC2C(z_c2c_plan,
			    (hipfftComplex *)zfft_grid->data,
			    (hipfftComplex *)zfft_grid->data, HIPFFT_BACKWARD));
    zfft_grid->transpose_xyz_yzx(xyfft_grid);
    cufftCheck(hipfftExecC2R(xy_c2r_plan,
			    (hipfftComplex *)xyfft_grid->data,
			    (hipfftReal *)xyfft_grid->data));
  } else if (fft_type == BOX) {
    cufftCheck(hipfftExecC2R(c2r_plan,
			    (hipfftComplex *)fft_grid->data,
			    (hipfftReal *)fft_grid->data));
  }

}

//
// Sets Bspline order
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::set_order(int order) {
  this->order = order;
  calc_prefac();
}

void dftmod(double *bsp_mod, const double *bsp_arr, const int nfft) {

  const double rsmall = 1.0e-10;
  double nfftr = (2.0*3.14159265358979323846)/(double)nfft;

  for (int k=1;k <= nfft;k++) {
    double sum1 = 0.0;
    double sum2 = 0.0;
    double arg1 = (k-1)*nfftr;
    for (int j=1;j < nfft;j++) {
      double arg = arg1*(j-1);
      sum1 += bsp_arr[j-1]*cos(arg);
      sum2 += bsp_arr[j-1]*sin(arg);
    }
    bsp_mod[k-1] = sum1*sum1 + sum2*sum2;
  }

  for (int k=1;k <= nfft;k++)
    if (bsp_mod[k-1] < rsmall)
      bsp_mod[k-1] = 0.5*(bsp_mod[k-1-1] + bsp_mod[k+1-1]);

  for (int k=1;k <= nfft;k++)
    bsp_mod[k-1] = 1.0/bsp_mod[k-1];

}

void fill_bspline_host(const int order, const double w, double *array, double *darray) {

  //--- do linear case
  array[order-1] = 0.0;
  array[2-1] = w;
  array[1-1] = 1.0 - w;

  //--- compute standard b-spline recursion
  for (int k=3;k <= order-1;k++) {
    double div = 1.0 / (double)(k-1);
    array[k-1] = div*w*array[k-1-1];
    for (int j=1;j <= k-2;j++)
      array[k-j-1] = div*((w+j)*array[k-j-1-1] + (k-j-w)*array[k-j-1]);
    array[1-1] = div*(1.0-w)*array[1-1];
  }

  //--- perform standard b-spline differentiation
  darray[1-1] = -array[1-1];
  for (int j=2;j <= order;j++)
    darray[j-1] = array[j-1-1] - array[j-1];

  //--- one more recursion
  int k = order;
  double div = 1.0 / (double)(k-1);
  array[k-1] = div*w*array[k-1-1];
  for (int j=1;j <= k-2;j++)
    array[k-j-1] = div*((w+j)*array[k-j-1-1] + (k-j-w)*array[k-j-1]);

  array[1-1] = div*(1.0-w)*array[1-1];

}

//
// Calculates (prefac_x, prefac_y, prefac_z)
// NOTE: This calculation is done on the CPU since it is only done infrequently
//
template <typename AT, typename CT, typename CT2>
void CudaPMERecip<AT, CT, CT2>::calc_prefac() {
  
  int max_nfft = max(nfftx, max(nffty, nfftz));
  double *bsp_arr = new double[max_nfft];
  double *bsp_mod = new double[max_nfft];
  double *array = new double[order];
  double *darray = new double[order];
  CT *h_prefac_x = new CT[nfftx];
  CT *h_prefac_y = new CT[nffty];
  CT *h_prefac_z = new CT[nfftz];

  fill_bspline_host(order, 0.0, array, darray);
  for (int i=0;i < max_nfft;i++) bsp_arr[i] = 0.0;
  for (int i=2;i <= order+1;i++) bsp_arr[i-1] = array[i-1-1];

  dftmod(bsp_mod, bsp_arr, nfftx);
  for (int i=0;i < nfftx;i++) h_prefac_x[i] = (CT)bsp_mod[i];

  dftmod(bsp_mod, bsp_arr, nffty);
  for (int i=0;i < nffty;i++) h_prefac_y[i] = (CT)bsp_mod[i];

  dftmod(bsp_mod, bsp_arr, nfftz);
  for (int i=0;i < nfftz;i++) h_prefac_z[i] = (CT)bsp_mod[i];

  copy_HtoD<CT>(h_prefac_x, prefac_x, nfftx);
  copy_HtoD<CT>(h_prefac_y, prefac_y, nffty);
  copy_HtoD<CT>(h_prefac_z, prefac_z, nfftz);

  delete [] bsp_arr;
  delete [] bsp_mod;
  delete [] array;
  delete [] darray;
  delete [] h_prefac_x;
  delete [] h_prefac_y;
  delete [] h_prefac_z;
}

//
// Explicit instances of CudaPMERecip
//
template class CudaPMERecip<long long int, float, float2>;
template class CudaPMERecip<int, float, float2>;
template void CudaPMERecip<int, float, float2>::gather_force<float>(const float4 *xyzq, const int ncoord,
								    const double* recip,
								    const int stride, float* force);
template void CudaPMERecip<int, float, float2>::gather_force<long long int>(const float4 *xyzq, const int ncoord,
									    const double* recip,
									    const int stride, long long int* force);
template void CudaPMERecip<int, float, float2>::gather_force<float3>(const float4 *xyzq, const int ncoord,
								     const double* recip,
								     const int stride, float3* force);

